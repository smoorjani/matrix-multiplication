#include "hip/hip_runtime.h"
// Taken from https://github.com/salehjg/batch-matmul-cuda

#include <stdio.h>
#include <assert.h>
#include <iostream>
#include <numeric>
#include <stdlib.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "Utilities.cuh"


#define CUDA_ERROR_CHECK

#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )


// C = AB
template <int BLOCK_SIZE>
__global__ void kernel_batched_matmul(
		const float * matA,
		const float * matB,
		float * matC,
		int dim0,
		int dim1A, int dim2A,
		int dim1B, int dim2B,
		int dim1C, int dim2C){
	extern __shared__ float smem[];

	const unsigned int len_subA = BLOCK_SIZE * dim2A,len_subB = BLOCK_SIZE * dim1B; //len of sub matrices of A and B.
	const unsigned long
		len_A = dim0*dim1A*dim2A,
		len_B = dim0*dim1B*dim2B,
		len_C = dim0*dim1C*dim2C;
	const unsigned long
		len_A_signleBatch = dim1A*dim2A,
		len_B_signleBatch = dim1B*dim2B,
		len_C_signleBatch = dim1C*dim2C;
	const unsigned int BLOCKSIZE_P2 = BLOCK_SIZE*BLOCK_SIZE;

    // Block index
    unsigned int bx = blockIdx.x; // mapped to the sub-matrices of output
    unsigned int by = blockIdx.y; // mapped to the sub-matrices of output
    unsigned int bz = blockIdx.z; // batch index

    // Thread index
    unsigned int  tx = threadIdx.x;
    unsigned int  ty = threadIdx.y;

    unsigned int  c_pos_x, c_pos_y;
    c_pos_x = bx*BLOCK_SIZE + tx;
    c_pos_y = by*BLOCK_SIZE + ty;

    unsigned long gidx1,gidx2;
    unsigned int _d1,_d2;


	unsigned long offsetA = (by * BLOCK_SIZE) * dim2A;
	unsigned long offsetB = (bx * BLOCK_SIZE); //first row (d1=0)

	// Load sub matrices from global memory into shared memory

	unsigned long idxA, idxB;
	idxA = ty* BLOCK_SIZE + tx;
	idxB = ty* BLOCK_SIZE + tx;

	while(idxA < len_subA){//Block-stride loop
		gidx1 = offsetA + idxA;
		if(idxA < len_subA && gidx1 < len_A) {
			smem[idxA] = matA[bz * len_A_signleBatch + gidx1];
		}else{
			smem[idxA] = 0;
		}
		idxA += BLOCKSIZE_P2;
	}

	///TODO: It might be better to store transposed subMatB in shared memory to avoid shared memory read conflict.
	///      But then we might get shared memory write conflict. (?)
	while(idxB < len_subB ){//Block-stride loop
		//gidx2 = offsetB + (bx*BLOCK_SIZE)*dim2B + (idxB % dim2B);
		_d2 = idxB%BLOCK_SIZE;
		_d1 = (idxB/BLOCK_SIZE);
		gidx2 = offsetB + _d1*dim2B + _d2;
		if(idxB < len_subB && _d1<dim1B && _d2<dim2B){
			smem[len_subA+idxB] = matB[bz * len_B_signleBatch +gidx2];
		}else{
			smem[len_subA+idxB] = 0;
		}
		idxB += BLOCKSIZE_P2;
	}

	__syncthreads();

    	// Multiply and add each result to produce output element of current thread in the thread block.
    if(c_pos_x<dim2C && c_pos_y<dim1C){
    	unsigned long idx = ty* BLOCK_SIZE + tx;
    	float output_element = 0.0f;

    	//dim2A=dim1B is common equal dimension of 2 matrices  --- block-stride loop
    	for (int k = 0; k < dim2A; k++) {
    		output_element += smem[ty*dim2A+k] * smem[len_subA+ k*BLOCK_SIZE+tx];
    	}

    	///TODO: Check matC index to not to exceed the len of matC!
    	matC[bz * len_C_signleBatch + c_pos_y*dim2C + c_pos_x] = output_element;

    }
}

void naive_batched_matmul(torch::Tensor d_A, torch::Tensor d_B,
            torch::Tensor d_C, int a_rows, int a_cols, int b_rows,
            int b_cols, int batch_dim) {

    float *d_A_arr = d_A.data_ptr<float>();
    float *d_B_arr = d_B.data_ptr<float>();
    float *d_C_arr = d_C.data_ptr<float>();

	const int BLOCK_DIM = 6;
	dim3 blocksize(BLOCK_DIM,BLOCK_DIM,1);
	dim3 gridsize(0,0,0);
	gridsize.x = (b_cols + BLOCK_DIM-1)/BLOCK_DIM;
	gridsize.y = (a_rows + BLOCK_DIM-1)/BLOCK_DIM;
	gridsize.z = (batch_dim);
	unsigned long sharedmemsize = (BLOCK_DIM*a_cols + BLOCK_DIM* b_rows)*sizeof(float);
	printf("@batched_matmul:\n");
	printf("\tBLOCK:(%d, %d)\n",blocksize.x,blocksize.y);
	printf("\t GRID:(%d, %d, %d)\n",gridsize.x,gridsize.y,gridsize.z);
	printf("\t SHARED: %d Bytes\n",sharedmemsize);

	if(BLOCK_DIM==6){
		kernel_batched_matmul<6> <<<gridsize, blocksize, sharedmemsize>>>(
				d_A_arr,
				d_B_arr,
				d_C_arr,
				batch_dim,

				a_rows, //hA
				a_cols, //wA

				b_rows, //hA
				b_cols, //wA

				a_rows,
				b_cols);
		CudaCheckError();
	}else{
		printf("ERR@batched_matmul: UNDEFINED BLOCK_DIM.\n"); return;
	}

}
