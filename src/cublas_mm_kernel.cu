#ifndef __CUBLAS_MM_KERNEL_H__
#define __CUBLAS_MM_KERNEL_H__

#include <iostream>
#include "Utilities.cuh"

#include <hip/hip_runtime.h>
#include <hipblas.h>

void cublas_mm_wrapper(hipblasHandle_t handle,
                       float *d_A, float *d_B, float *d_C,
                       int m, int k, int n) {
    hipMemset(d_C, 0, m * n * sizeof(float));
    
    float alpha = 1.0;
    float beta = 0.0;
    hipblasStatus_t status = hipblasSgemm(
        handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
        n, m, k, &alpha,
        d_B, n,
        d_A, k, &beta,
        d_C, n);

    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        std::cerr << "Kernel execution error.";
    }

}

#endif // __CUBLAS_MM_KERNEL_H__
