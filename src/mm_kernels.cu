#include "hip/hip_runtime.h"
#ifndef __MM_KERNEL_H__
#define __MM_KERNEL_H__

#include <iostream>
#include <torch/extension.h>
#include "Utilities.cuh"

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsparse.h>
#include <hipblaslt.h>

#include <sys/time.h>
typedef unsigned long long timestamp_t;

static timestamp_t get_timestamp () {
    struct timeval now;
    gettimeofday (&now, NULL);
    return  now.tv_usec + (timestamp_t)now.tv_sec * 1000000;
}

#define NUM_THREADS (64)

__global__ void packed_1d_accessor_kernel(
    // figure out if this is coalesced
    torch::PackedTensorAccessor32<float, 2> accessor,
    float* trace
) {
  int n_rows = accessor.size(0);
  int n_cols = accessor.size(1);
    
  int idx_per_row = (int) ((float) (n_cols + NUM_THREADS - 1) / NUM_THREADS);
  // find thread id, row = threadid/64, col = threadid%64
  int threadId = threadIdx.x + blockDim.x * blockIdx.x;
  int row = 0, col = 0;

  if (n_cols >= NUM_THREADS) {
    int idx = threadId / NUM_THREADS;
    int off = threadId % NUM_THREADS;

    row = idx;
    col = row * (idx_per_row - 1) + off;
  } else {
    row = threadId / n_cols;
    col = threadId % n_cols;
  }

  if (row >= n_rows || col >= n_cols) {
    return;
  }
  trace[row * n_cols + col] = accessor[row][col];
}

__global__ void packed_1d_setter_kernel(
    // figure out if this is coalesced
    torch::PackedTensorAccessor32<float, 2> accessor,
    float* trace
) {
  int n_rows = accessor.size(0);
  int n_cols = accessor.size(1);
    
  int idx_per_row = (int) ((float) (n_cols + NUM_THREADS - 1) / NUM_THREADS);
  // find thread id, row = threadid/64, col = threadid%64
  int threadId = threadIdx.x + blockDim.x * blockIdx.x;
  int row = 0, col = 0;

  if (n_cols >= NUM_THREADS) {
    int idx = threadId / NUM_THREADS;
    int off = threadId % NUM_THREADS;

    row = idx;
    col = row * (idx_per_row - 1) + off;
  } else {
    row = threadId / n_cols;
    col = threadId % n_cols;
  }

  if (row >= n_rows || col >= n_cols) {
    return;
  }
  
  accessor[row][col] = trace[row * n_cols + col];
}

void cublas_mm_wrapper(hipblasHandle_t handle,
                       torch::Tensor d_A, torch::Tensor d_B, torch::Tensor d_C,
                       int a_rows, int b_rows, int b_cols) {

    /*
    size_t a_size = sizeof(float) * a_rows * b_rows;
    size_t b_size = sizeof(float) * b_rows * b_cols;
    size_t c_size = sizeof(float) * a_rows * b_cols;

    float *d_A_arr, *d_B_arr, *d_C_arr;
    gpuErrchk(hipMalloc((void **)&d_A_arr, a_size));
    gpuErrchk(hipMalloc((void **)&d_B_arr, b_size));
    gpuErrchk(hipMalloc((void **)&d_C_arr, c_size));

    auto A_accessor = d_A.packed_accessor32<float,2>();
    auto B_accessor = d_B.packed_accessor32<float,2>();
    auto C_accessor = d_C.packed_accessor32<float,2>();
 
    dim3 thread_per_block(NUM_THREADS);
    dim3 A_blocks_per_grid((a_rows * b_rows + NUM_THREADS - 1)/NUM_THREADS);
    dim3 B_blocks_per_grid((b_rows * b_cols + NUM_THREADS - 1)/NUM_THREADS);
    dim3 C_blocks_per_grid((a_rows * b_cols + NUM_THREADS - 1)/NUM_THREADS);

    packed_1d_accessor_kernel<<<A_blocks_per_grid, thread_per_block>>>(A_accessor, d_A_arr);   
    packed_1d_accessor_kernel<<<B_blocks_per_grid, thread_per_block>>>(B_accessor, d_B_arr);
    */
    
    float *d_A_arr = d_A.data_ptr<float>();
    float *d_B_arr = d_B.data_ptr<float>();
    float *d_C_arr = d_C.data_ptr<float>();

    float alpha = 1.0;
    float beta = 0.0;
    hipblasStatus_t status = hipblasSgemm(
        handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
        b_cols, a_rows, b_rows, &alpha,
        d_B_arr, b_cols,
        d_A_arr, b_rows, &beta,
        d_C_arr, b_cols);

    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        std::cerr << "Kernel execution error.";
    }
    
    /*
    packed_1d_setter_kernel<<<C_blocks_per_grid, thread_per_block>>>(C_accessor, d_C_arr);
    
    gpuErrchk(hipFree(d_A_arr));
    gpuErrchk(hipFree(d_B_arr));
    gpuErrchk(hipFree(d_C_arr));
    */
}

__global__ void packed_2d_accessor_kernel(
    // figure out if this is coalesced
    torch::PackedTensorAccessor32<float, 3> accessor,
    float* trace
) {
  int batch_size = accessor.size(0);
  int n_rows = accessor.size(1);
  int n_cols = accessor.size(2);
    
  int idx_per_row = (int) ((float) (n_cols + NUM_THREADS - 1) / NUM_THREADS);
  // find thread id, row = threadid/64, col = threadid%64
  int threadId = threadIdx.x + blockDim.x * blockIdx.x;
  int batch = 0, row = 0, col = 0;

  if (n_cols >= NUM_THREADS) {
    int idx = threadId / NUM_THREADS;
    int off = threadId % NUM_THREADS;

    batch = idx / (idx_per_row * n_rows);
    row = (idx / idx_per_row) % n_rows;
    col = (idx % idx_per_row) * NUM_THREADS + off;
  } else {
    batch = threadId / (n_rows * n_cols);
    row = (threadId % (n_rows * n_cols))/ n_cols;
    col = threadId % n_cols;
  }

  if (batch >= batch_size || row >= n_rows || col >= n_cols) {
    return;
  }
  
  trace[batch * n_rows * n_cols + row * n_cols + col] = accessor[batch][row][col];
}

__global__ void packed_2d_accessor_kernel_combined(
    // figure out if this is coalesced
    torch::PackedTensorAccessor32<float, 3> a_accessor,
    torch::PackedTensorAccessor32<float, 3> b_accessor,
    float* a_trace,
    float* b_trace
) {
  int batch_size = a_accessor.size(0);
  int a_rows = a_accessor.size(1);
  int a_cols = a_accessor.size(2);
  int b_rows = b_accessor.size(1);
  int b_cols = b_accessor.size(2);
    
  int a_idx_per_row = (int) ((float) (a_cols + NUM_THREADS - 1) / NUM_THREADS);
  int b_idx_per_row = (int) ((float) (b_cols + NUM_THREADS - 1) / NUM_THREADS);
  // find thread id, row = threadid/64, col = threadid%64
  int threadId = threadIdx.x + blockDim.x * blockIdx.x;
  int a_batch = 0, a_row = 0, a_col = 0;
  int b_batch = 0, b_row = 0, b_col = 0;

  if (a_cols >= NUM_THREADS) {
    int idx = threadId / NUM_THREADS;
    int off = threadId % NUM_THREADS;

    a_batch = idx / (a_idx_per_row * a_rows);
    a_row = (idx / a_idx_per_row) % a_rows;
    a_col = (idx % a_idx_per_row) * NUM_THREADS + off;
  } else {
    a_batch = threadId / (a_rows * a_cols);
    a_row = (threadId % (a_rows * a_cols))/ a_cols;
    a_col = threadId % a_cols;
  }

  if (b_cols >= NUM_THREADS) {
    int idx = threadId / NUM_THREADS;
    int off = threadId % NUM_THREADS;

    b_batch = idx / (b_idx_per_row * b_rows);
    b_row = (idx / b_idx_per_row) % b_rows;
    b_col = (idx % b_idx_per_row) * NUM_THREADS + off;
  } else {
    b_batch = threadId / (b_rows * b_cols);
    b_row = (threadId % (b_rows * b_cols))/ b_cols;
    b_col = threadId % b_cols;
  }

  if (a_batch < batch_size && a_row < a_rows && a_col < a_cols) {
    a_trace[a_batch * a_rows * a_cols + a_row * a_cols + a_col] = a_accessor[a_batch][a_row][a_col];
  }

  if (b_batch < batch_size && b_row < b_rows && b_col < b_cols) {
    b_trace[b_batch * b_rows * b_cols + b_row * b_cols + b_col] = b_accessor[b_batch][b_row][b_col];
  }
 
}


__global__ void packed_2d_setter_kernel(
    torch::PackedTensorAccessor32<float, 3> accessor,
    float* trace
) {
  int batch_size = accessor.size(0);
  int n_rows = accessor.size(1);
  int n_cols = accessor.size(2);
  int idx_per_row = (int) ((float) (n_cols + NUM_THREADS - 1) / NUM_THREADS);
  // find thread id, row = threadid/64, col = threadid%64
  int threadId = threadIdx.x + blockDim.x * blockIdx.x;
  int batch= 0, row = 0, col = 0;

  if (n_cols >= NUM_THREADS) {
    int idx = threadId / NUM_THREADS;
    int off = threadId % NUM_THREADS;

    batch = idx / (idx_per_row * n_rows);
    row = (idx / idx_per_row) % n_rows;
    col = (idx % idx_per_row) * NUM_THREADS + off;
  } else {
    batch = threadId / (n_rows * n_cols);
    row = (threadId % (n_rows * n_cols))/ n_cols;
    col = threadId % n_cols;
  }

  if (batch >= batch_size || row >= n_rows || col >= n_cols) {
    return;
  }

  accessor[batch][row][col] = trace[batch * n_rows * n_cols + row * n_cols + col];
}

// https://stackoverflow.com/questions/23743384/how-performing-multiple-matrix-multiplications-in-cuda/23743838#23743838

void cublas_bmm_wrapper(hipblasHandle_t handle,
               torch::Tensor d_A, torch::Tensor d_B, torch::Tensor d_C,
               size_t a_rows, size_t b_cols, size_t b_rows,
               size_t batch_dim) {
    
    // ==============
    timestamp_t t0 = get_timestamp();
    float *d_A_arr = d_A.data_ptr<float>();
    float *d_B_arr = d_B.data_ptr<float>();
    float *d_C_arr = d_C.data_ptr<float>();
    /*
    size_t a_size = sizeof(float) * a_rows * b_rows;
    size_t b_size = sizeof(float) * b_rows * b_cols;
    size_t c_size = sizeof(float) * a_rows * b_cols;
    
    // setting up device arrays for bmm
    float *d_A_arr, *d_B_arr, *d_C_arr;
    gpuErrchk(hipMalloc(&d_A_arr, batch_dim * a_size));
    gpuErrchk(hipMalloc(&d_B_arr, batch_dim * b_size));
    gpuErrchk(hipMalloc(&d_C_arr, batch_dim * c_size));
    
    // creating accessors for tensors
    auto A_accessor = d_A.packed_accessor32<float,3>();
    auto B_accessor = d_B.packed_accessor32<float,3>();
    auto C_accessor = d_C.packed_accessor32<float,3>();

    // const int num_streams = 16;
    // hipStream_t streams[num_streams];
    // execute 1 time with a_rows threads
    // << blocks_per_grid/NUM_THREADS, NUM_THREADS, 0, streams[i]>>
    dim3 thread_per_block(NUM_THREADS);
    dim3 C_blocks_per_grid((batch_dim * a_rows * b_cols + NUM_THREADS - 1)/NUM_THREADS);
    dim3 AB_blocks_per_grid((batch_dim * a_rows * b_rows + batch_dim * a_rows * b_cols + NUM_THREADS - 1)/NUM_THREADS);
    */
    timestamp_t t1 = get_timestamp();
    double secs = (t1 - t0) / 1000000.0L;
    printf("Preprocessing: %f\n", secs);
    /*
    // ==============
    t0 = get_timestamp();
    packed_2d_accessor_kernel_combined<<<AB_blocks_per_grid, thread_per_block>>>(A_accessor, B_accessor, d_A_arr, d_B_arr);
    gpuErrchk(hipDeviceSynchronize());
    t1 = get_timestamp();
    secs = (t1 - t0) / 1000000.0L;
    printf("Accessor Kernel: %f\n", secs);
    // ==============
    */
    t0 = get_timestamp();
    const float alpha = 1.0f, beta = 0.0f;

    hipblasStatus_t status = hipblasSgemmStridedBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N
                                       , b_cols, a_rows, b_rows
                                       , &alpha, d_B_arr, b_cols, b_rows * b_cols
                                       , d_A_arr, b_rows, a_rows * b_rows
                                       , &beta, d_C_arr, b_cols, a_rows * b_cols
                                       , batch_dim);
    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        std::cerr << "Kernel execution error.";
    }

    gpuErrchk(hipDeviceSynchronize());
    t1 = get_timestamp();
    secs = (t1 - t0) / 1000000.0L;
    printf("Batch GEMM: %f\n", secs);
    /*
    // ==============
    t0 = get_timestamp();
    packed_2d_setter_kernel<<<C_blocks_per_grid, thread_per_block>>>(C_accessor, d_C_arr);
    gpuErrchk(hipDeviceSynchronize());
    t1 = get_timestamp();
    secs = (t1 - t0) / 1000000.0L;
    printf("Setter: %f\n", secs);
    // ==============
    t0 = get_timestamp();
    gpuErrchk(hipFree(d_A_arr));
    gpuErrchk(hipFree(d_B_arr));
    gpuErrchk(hipFree(d_C_arr));
    t1 = get_timestamp();
    secs = (t1 - t0) / 1000000.0L;
    printf("Freeing Memory: %f\n", secs);
    // ==============
    */
}

__global__ void packed_2d_accessor_kernel_4d(
    // figure out if this is coalesced
    torch::PackedTensorAccessor32<float, 4> accessor,
    float* trace
) {
  // turns a 4d matrix into a 2d array of [batch_size][matrix_idx]
  // batch size spans the first two dimensions
  // matrix idx spans the last two (rows/cols)
  int batch_dim1 = accessor.size(0);
  int batch_dim2 = accessor.size(1);
  int n_rows = accessor.size(2);
  int n_cols = accessor.size(3);
    
  int idx_per_row = (int) ((float) (n_cols + NUM_THREADS - 1) / NUM_THREADS);
  // find thread id, row = threadid/64, col = threadid%64
  int threadId = threadIdx.x + blockDim.x * blockIdx.x;
  int d_1 = 0, d_2 = 0, row = 0, col = 0;

  if (n_cols >= NUM_THREADS) {
    int idx = threadId / NUM_THREADS;
    int off = threadId % NUM_THREADS;

    d_1 = idx / (idx_per_row * n_rows * batch_dim2);
    d_2 = (idx / (idx_per_row * n_rows)) % batch_dim2;
    row = (idx / idx_per_row) % n_rows;
    col = (idx % idx_per_row) * NUM_THREADS + off;
  } else {
    d_1 = threadId / (batch_dim2 * n_rows * n_cols);
    d_2 = (threadId % (batch_dim2 * n_rows * n_cols))/ (n_rows * n_cols);
    row = (threadId % (n_rows * n_cols))/ n_cols;
    col = threadId % n_cols;
  }

  if (d_1 >= batch_dim1 || d_2 >= batch_dim2 || row >= n_rows || col >= n_cols) {
    return;
  }

  trace[threadId] = accessor[d_1][d_2][row][col];
}

__global__ void packed_2d_setter_kernel_4d(
    // figure out if this is coalesced
    torch::PackedTensorAccessor32<float, 4> accessor,
    float* trace
) {
  // turns a 4d matrix into a 2d array of [batch_size][matrix_idx]
  // batch size spans the first two dimensions
  // matrix idx spans the last two (rows/cols)
  int batch_dim1 = accessor.size(0);
  int batch_dim2 = accessor.size(1);
  int n_rows = accessor.size(2);
  int n_cols = accessor.size(3);
    
  int idx_per_row = (int) ((float) (n_cols + NUM_THREADS - 1) / NUM_THREADS);
  // find thread id, row = threadid/64, col = threadid%64
  int threadId = threadIdx.x + blockDim.x * blockIdx.x;
  int d_1 = 0, d_2 = 0, row = 0, col = 0;

  if (n_cols >= NUM_THREADS) {
    int idx = threadId / NUM_THREADS;
    int off = threadId % NUM_THREADS;


    d_1 = idx / (idx_per_row * n_rows * batch_dim2);
    d_2 = (idx / (idx_per_row * n_rows)) % batch_dim2;
    row = (idx / idx_per_row) % n_rows;
    col = (idx % idx_per_row) * NUM_THREADS + off;
  } else {
    d_1 = threadId / (batch_dim2 * n_rows * n_cols);
    d_2 = (threadId % (batch_dim2 * n_rows * n_cols))/ (n_rows * n_cols);
    row = (threadId % (n_rows * n_cols))/ n_cols;
    col = threadId % n_cols;
  }

  if (d_1 >= batch_dim1 || d_2 >= batch_dim2 || row >= n_rows || col >= n_cols) {
    return;
  }
  
  accessor[d_1][d_2][row][col] = trace[threadId];
}

void cublas_4d_bmm_wrapper(hipblasHandle_t handle,
               torch::Tensor d_A, torch::Tensor d_B, torch::Tensor d_C,
               size_t a_rows, size_t b_cols, size_t b_rows,
               size_t batch_dim1, size_t batch_dim2) {

    /*
    size_t a_size = sizeof(float) * a_rows * b_rows;
    size_t b_size = sizeof(float) * b_rows * b_cols;
    size_t c_size = sizeof(float) * a_rows * b_cols;

    // setting up device arrays for bmm
    float *d_A_arr, *d_B_arr, *d_C_arr;
    gpuErrchk(hipMalloc(&d_A_arr, batch_dim1 * batch_dim2 * a_size));
    gpuErrchk(hipMalloc(&d_B_arr, batch_dim1 * batch_dim2 * b_size));
    gpuErrchk(hipMalloc(&d_C_arr, batch_dim1 * batch_dim2 * c_size));

    auto A_accessor = d_A.packed_accessor32<float,4>();
    auto B_accessor = d_B.packed_accessor32<float,4>();
    auto C_accessor = d_C.packed_accessor32<float,4>();

    // execute 1 time with a_rows threads
    dim3 thread_per_block(NUM_THREADS);
    dim3 A_blocks_per_grid((batch_dim1 * batch_dim2 * a_rows * b_rows + NUM_THREADS - 1)/NUM_THREADS);
    dim3 B_blocks_per_grid((batch_dim1 * batch_dim2 * b_rows * b_cols + NUM_THREADS - 1)/NUM_THREADS);
    dim3 C_blocks_per_grid((batch_dim1 * batch_dim2 * a_rows * b_cols + NUM_THREADS - 1)/NUM_THREADS);

    // <<<total threads/64 ,64>>>>
    packed_2d_accessor_kernel_4d<<<A_blocks_per_grid, thread_per_block>>>(A_accessor, d_A_arr);
    packed_2d_accessor_kernel_4d<<<B_blocks_per_grid, thread_per_block>>>(B_accessor, d_B_arr);
 
    gpuErrchk(hipDeviceSynchronize());
    */

    float *d_A_arr = d_A.data_ptr<float>();
    float *d_B_arr = d_B.data_ptr<float>();
    float *d_C_arr = d_C.data_ptr<float>();

    const float alpha = 1.0f, beta = 0.0f;
       
    hipblasStatus_t status = hipblasSgemmStridedBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N
                                       , b_cols, a_rows, b_rows
                                       , &alpha, d_B_arr, b_cols, b_rows * b_cols
                                       , d_A_arr, b_rows, a_rows * b_rows
                                       , &beta, d_C_arr, b_cols, a_rows * b_cols
                                       , batch_dim1 * batch_dim2);
    
    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        std::cerr << "Kernel execution error.";
    }

    /*
    gpuErrchk(hipDeviceSynchronize());

    packed_2d_setter_kernel_4d<<<C_blocks_per_grid, thread_per_block>>>(C_accessor, d_C_arr);
    
    gpuErrchk(hipFree(d_A_arr));
    gpuErrchk(hipFree(d_B_arr));
    gpuErrchk(hipFree(d_C_arr));
    */
}

/*
cuSPARSE Kernels
https://stackoverflow.com/questions/29688627/sparse-matrix-matrix-multiplication-in-cuda-using-cusparse

Sparse (CSR) * Dense matmul
A * B = C

(m x k) * (k * n) = (m x n)
note: row_ind.len = lda + 1
*/
void cusparse_mm_wrapper(hipsparseHandle_t handle,
                         double *d_A, int *d_A_ColIndices, int *d_A_RowIndices,
                         int nnzA, int A_rowptr_size,
                         double *d_B_dense, int B_rows, int B_cols,
                         double *d_C_dense)
{
    // Initialize cuSPARSE
    // hipsparseHandle_t handle;
    // cusparseSafeCall(hipsparseCreate(&handle));
    const int m = A_rowptr_size - 1;
    const int k = B_rows;
    const int n = B_cols;

    // Descriptor for sparse matrix A
    hipsparseMatDescr_t descrA;
    cusparseSafeCall(hipsparseCreateMatDescr(&descrA));
    cusparseSafeCall(hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL));
    cusparseSafeCall(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ONE));
    // Descriptor for sparse matrix B
    hipsparseMatDescr_t descrB;
    cusparseSafeCall(hipsparseCreateMatDescr(&descrB));
    cusparseSafeCall(hipsparseSetMatType(descrB, HIPSPARSE_MATRIX_TYPE_GENERAL));
    cusparseSafeCall(hipsparseSetMatIndexBase(descrB, HIPSPARSE_INDEX_BASE_ONE));
    // Descriptor for sparse matrix C
    hipsparseMatDescr_t descrC;
    cusparseSafeCall(hipsparseCreateMatDescr(&descrC));
    cusparseSafeCall(hipsparseSetMatType(descrC, HIPSPARSE_MATRIX_TYPE_GENERAL));
    cusparseSafeCall(hipsparseSetMatIndexBase(descrC, HIPSPARSE_INDEX_BASE_ONE));

    int nnzB = 0; //   Number of nonzero elements in dense matrix B
    // Device side number of nonzero elements per row of matrix B
    int *d_nnzPerVectorB;
    gpuErrchk(hipMalloc(&d_nnzPerVectorB, k * sizeof(*d_nnzPerVectorB)));
    cusparseSafeCall(hipsparseDnnz(handle, HIPSPARSE_DIRECTION_ROW, k, n, descrB, d_B_dense, k, d_nnzPerVectorB, &nnzB));

    // Device side sparse matrix B
    double *d_B;
    gpuErrchk(hipMalloc(&d_B, nnzB * sizeof(*d_B)));
    int *d_B_RowIndices;
    gpuErrchk(hipMalloc(&d_B_RowIndices, (k + 1) * sizeof(*d_B_RowIndices)));
    int *d_B_ColIndices;
    gpuErrchk(hipMalloc(&d_B_ColIndices, nnzB * sizeof(*d_B_ColIndices)));
    // Dense B to Sparse B
    cusparseSafeCall(hipsparseDdense2csr(handle, k, n, descrB, d_B_dense, k, d_nnzPerVectorB, d_B, d_B_RowIndices, d_B_ColIndices));

    // Device side sparse matrix C
    int *d_C_RowIndices;
    gpuErrchk(hipMalloc(&d_C_RowIndices, (m + 1) * sizeof(*d_C_RowIndices)));

    // Performing the matrix - matrix multiplication
    int baseC, nnzC = 0;
    // nnzTotalDevHostPtr points to host memory
    int *nnzTotalDevHostPtr = &nnzC;

    cusparseSafeCall(hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST));

    //cusparseSafeCall(hipsparseXcsrgemmNnz(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, n, k, descrA, nnzA,
    //                                     d_A_RowIndices, d_A_ColIndices, descrB, nnzB, d_B_RowIndices, d_B_ColIndices, descrC, d_C_RowIndices,
    //                                     nnzTotalDevHostPtr));
    if (nnzTotalDevHostPtr != NULL)
        nnzC = *nnzTotalDevHostPtr;
    else
    {
        hipMemcpy(&nnzC, d_C_RowIndices + m, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&baseC, d_C_RowIndices, sizeof(int), hipMemcpyDeviceToHost);
        nnzC -= baseC;
    }

    // device side sparse matrix C
    double *d_C;
    gpuErrchk(hipMalloc(&d_C, nnzC * sizeof(double)));
    int *d_C_ColIndices;
    gpuErrchk(hipMalloc(&d_C_ColIndices, nnzC * sizeof(int)));

    //cusparseSafeCall(hipsparseDcsrgemm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, n, k, descrA, nnzA,
    //                                  d_A, d_A_RowIndices, d_A_ColIndices, descrB, nnzB, d_B, d_B_RowIndices, d_B_ColIndices, descrC,
    //                                  d_C, d_C_RowIndices, d_C_ColIndices));

    cusparseSafeCall(hipsparseDcsr2dense(handle, m, n, descrC, d_C, d_C_RowIndices, d_C_ColIndices, d_C_dense, m));

    hipFree(d_nnzPerVectorB);

    hipFree(d_B);
    hipFree(d_B_RowIndices);
    hipFree(d_B_ColIndices);

    hipFree(d_C);
    hipFree(d_C_RowIndices);
    hipFree(d_C_ColIndices);

    return;
}

void dense_to_csr(hipsparseHandle_t handle, 
                  double *d_A_dense, const int Nrows, const int Ncols,
                  double **d_A_val, int **d_A_colind, int **d_A_rowptr, int *nnzA)
{
    // Descriptor for sparse matrix A
    hipsparseMatDescr_t descrA;
    cusparseSafeCall(hipsparseCreateMatDescr(&descrA));
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);

    int nnz = 0;           //   Number of nonzero elements in dense matrix
    const int lda = Nrows; //   Leading dimension of dense matrix
    // Device side number of nonzero elements per row
    int *d_nnzPerVector;
    gpuErrchk(hipMalloc(&d_nnzPerVector, Nrows * sizeof(*d_nnzPerVector)));
    cusparseSafeCall(hipsparseDnnz(handle, HIPSPARSE_DIRECTION_ROW, Nrows, Ncols, descrA, d_A_dense, lda, d_nnzPerVector, &nnz));

    // Device side sparse matrix
    double *d_A;
    gpuErrchk(hipMalloc(&d_A, nnz * sizeof(*d_A)));
    int *d_A_RowIndices;
    gpuErrchk(hipMalloc(&d_A_RowIndices, (Nrows + 1) * sizeof(*d_A_RowIndices)));
    int *d_A_ColIndices;
    gpuErrchk(hipMalloc(&d_A_ColIndices, nnz * sizeof(*d_A_ColIndices)));

    cusparseSafeCall(hipsparseDdense2csr(handle, Nrows, Ncols, descrA, d_A_dense, lda, d_nnzPerVector, d_A, d_A_RowIndices, d_A_ColIndices));

    *d_A_val = d_A;
    *d_A_rowptr = d_A_RowIndices;
    *d_A_colind = d_A_ColIndices;
    *nnzA = nnz;

    gpuErrchk(hipFree(d_nnzPerVector));
    return;
}

int roundoff(int v, int d) {
    return (v + d - 1) / d * d;
}

void LtIgemmTensor(hipblasLtHandle_t ltHandle,
                   int m,
                   int n,
                   int k,
                   const float *A,
                   int lda,
                   const float *B,
                   int ldb,
                   float *C,
                   int ldc) {
    hipblasLtMatmulDesc_t matmulDesc = NULL;
    hipblasLtMatrixLayout_t Adesc = NULL, Bdesc = NULL, Cdesc = NULL;
    int32_t alpha = 1, beta = 0;
    hipblasOperation_t opTranspose = HIPBLAS_OP_T;

    // tensor op igemm kernels require specialized memory order of data
    hipblasLtMatrixTransformDesc_t transformDesc = NULL;
    float *Atransform = NULL, *Btransform = NULL;
    float *Ctransform                   = NULL;
    hipblasLtMatrixLayout_t AtransformDesc = NULL, BtransformDesc = NULL, CtransformDesc = NULL;
    float transformAlpha = 1.0f, transformBeta = 0.0f;
    hipblasLtOrder_t order_COL32       = CUBLASLT_ORDER_COL32;
    hipblasLtOrder_t order_COL4_4R2_8C = CUBLASLT_ORDER_COL4_4R2_8C;

    int ldatransform = 32 * m;
    int ldbtransform = 32 * roundoff(n, 8);
    int ldctransform = 32 * m;

    checkCudaStatus(hipMalloc(reinterpret_cast<void**>(&Atransform), sizeof(float) * roundoff(k, 32) / 32 * ldatransform));
    checkCudaStatus(hipMalloc(reinterpret_cast<void**>(&Btransform), sizeof(float) * roundoff(k, 32) / 32 * ldbtransform));
    checkCudaStatus(hipMalloc(reinterpret_cast<void**>(&Ctransform), sizeof(float) * roundoff(n, 32) / 32 * ldctransform));

    checkCublasStatus(hipblasLtMatrixTransformDescCreate(&transformDesc, HIP_R_32F));

    //checkCublasStatus(hipblasLtMatmulDescCreate(&matmulDesc, HIP_R_32F));
    // tensor op igemm kernels only support NT gemm
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &opTranspose, sizeof(opTranspose)));

    // ---------------------------------------------------------------------------------------------
    // create descriptors for original matrices

    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_32F, m, k, lda));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_32F, k, n, ldb));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_32F, m, n, ldc));

    // ---------------------------------------------------------------------------------------------
    // create descriptors for transformed matrices

    checkCublasStatus(hipblasLtMatrixLayoutCreate(&AtransformDesc, HIP_R_32F, m, k, ldatransform));
    checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(AtransformDesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &order_COL32, sizeof(order_COL32)));

    // data memory order is set to CUBLASLT_ORDER_COL4_4R2_8C in order to achieve best performance on Turing devices.
    // for best performance on Ampere, consider setting the memory order to CUBLASLT_ORDER_COL32_2R_4R4.
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&BtransformDesc, HIP_R_32F, n, k, ldbtransform));
    checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(BtransformDesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &order_COL4_4R2_8C, sizeof(order_COL4_4R2_8C)));

    checkCublasStatus(hipblasLtMatrixLayoutCreate(&CtransformDesc, HIP_R_32F, m, n, ldctransform));
    checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(CtransformDesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &order_COL32, sizeof(order_COL32)));

    // ---------------------------------------------------------------------------------------------
    // transforms and computation

    checkCublasStatus(hipblasLtMatrixTransform(ltHandle, transformDesc, &transformAlpha, A, Adesc, &transformBeta, NULL, NULL, Atransform, AtransformDesc, 0));

    // B matrix is non-transposed, but transposed matrix is needed - add transpose operation in matrix transform.
    checkCublasStatus(hipblasLtMatrixTransformDescSetAttribute(transformDesc, HIPBLASLT_MATRIX_TRANSFORM_DESC_TRANSA, &opTranspose, sizeof(opTranspose)));

    checkCublasStatus(hipblasLtMatrixTransform(ltHandle, transformDesc, &transformAlpha, B, Bdesc, &transformBeta, NULL, NULL, Btransform, BtransformDesc, 0));

    // no need to transform C matrix as beta is assumed to be 0
    checkCublasStatus(hipblasLtMatmul(ltHandle,
                                     matmulDesc,
                                     &alpha,
                                     Atransform,
                                     AtransformDesc,
                                     Btransform,
                                     BtransformDesc,
                                     &beta,
                                     Ctransform,
                                     CtransformDesc,
                                     Ctransform,
                                     CtransformDesc,
                                     NULL,
                                     NULL,
                                     0,
                                     0));

    opTranspose = HIPBLAS_OP_N;
    checkCublasStatus(hipblasLtMatrixTransformDescSetAttribute(transformDesc, HIPBLASLT_MATRIX_TRANSFORM_DESC_TRANSA, &opTranspose, sizeof(opTranspose)));

    // transform outputs to COL order
    checkCublasStatus(hipblasLtMatrixTransform(ltHandle, transformDesc, &transformAlpha, Ctransform, CtransformDesc, &transformBeta, NULL, NULL, C, Cdesc, 0));

    // descriptors are no longer needed as all GPU work was already enqueued
    if (CtransformDesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(CtransformDesc));
    if (BtransformDesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(BtransformDesc));
    if (AtransformDesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(AtransformDesc));
    if (Cdesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Cdesc));
    if (Bdesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Bdesc));
    if (Adesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Adesc));
    if (matmulDesc) checkCublasStatus(hipblasLtMatmulDescDestroy(matmulDesc));
    if (transformDesc) checkCublasStatus(hipblasLtMatrixTransformDescDestroy(transformDesc));

    // wait until device is done before freeing transformed buffers
    checkCudaStatus(hipDeviceSynchronize());
    if (Ctransform) checkCudaStatus(hipFree(Ctransform));
    if (Btransform) checkCudaStatus(hipFree(Btransform));
    if (Atransform) checkCudaStatus(hipFree(Atransform));
}

#endif // __MM_KERNEL_H__
