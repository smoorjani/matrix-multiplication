#include "hip/hip_runtime.h"
#ifndef __MM_KERNEL_H__
#define __MM_KERNEL_H__

#include <iostream>
#include <torch/extension.h>
#include "Utilities.cuh"

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsparse.h>
#include <hipblaslt.h>

#include <sys/time.h>
typedef unsigned long long timestamp_t;

static timestamp_t get_timestamp () {
    struct timeval now;
    gettimeofday (&now, NULL);
    return  now.tv_usec + (timestamp_t)now.tv_sec * 1000000;
}

#define NUM_THREADS (64)

__global__ void dummyKernel()
{
    int tid = threadIdx.x+blockIdx.x*blockDim.x;
    printf("%d\n", tid);
}

void dummy_kernel_launch() {
    dim3 threads_per_block(NUM_THREADS);
    dim3 blocks_per_grid(NUM_THREADS);
    dummyKernel<<<blocks_per_grid, threads_per_block>>>();
    gpuErrchk(hipDeviceSynchronize());
}

void cublas_mm_wrapper(hipblasHandle_t handle,
                       torch::Tensor d_A, torch::Tensor d_B, torch::Tensor d_C,
                       int a_rows, int a_cols, int b_rows, int b_cols,
                       bool transa, bool transb) {

    float *d_A_arr = d_A.data_ptr<float>();
    float *d_B_arr = d_B.data_ptr<float>();
    float *d_C_arr = d_C.data_ptr<float>();

    printf("transa: %d, transb: %d\n", transa, transb);
    printf("a_rows: %d, b_rows: %d, b_cols: %d\n", a_rows, b_rows, b_cols);
    hipblasOperation_t trans_a = (!transb) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    hipblasOperation_t trans_b = (!transa) ? HIPBLAS_OP_N : HIPBLAS_OP_T;

    int m = b_cols;
    int n = a_rows;
    int k = b_rows;

    int ldb = b_cols;
    int lda = b_rows;
    int ldc = b_cols;

    if (transb && transa) {
        m = b_rows;
	k = b_cols;
	ldb = b_rows;
        lda = b_cols;
	ldc = lda;
    } else if (transa) {
        n = a_cols;
	m = b_cols;
	k = b_rows;
	lda = k;
        ldb = m;
	ldc = m;
    } else if (transb) {
        m = b_rows;
        n = a_cols;
	k = a_rows;
	ldb = n;
        lda = k;
	ldc = lda;
    }


    printf("m: %d, n: %d, k: %d\n", m, n, k);
    printf("lda: %d, ldb: %d, ldc: %d\n", lda, ldb, ldc);

    float alpha = 1.0;
    float beta = 0.0;
    hipblasStatus_t status = hipblasSgemm(
        handle, trans_a, trans_b,
        m, n, k, &alpha,
        d_B_arr, ldb,
        d_A_arr, lda, &beta,
        d_C_arr, ldc);

    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        std::cerr << "Kernel execution error.";
    }
}

__global__ void check_equal(float *d_Arr, float *h_Arr, size_t rows, size_t cols)
{
    int tid = threadIdx.x+blockIdx.x*blockDim.x;
    if (tid >= rows * cols) {
        return;
    }

    if (d_Arr[tid] == h_Arr[tid]) {
        printf("Equal %d\n", tid);
    } else {
        printf("Not Equal %d\n", tid);
    }
}
// https://stackoverflow.com/questions/23743384/how-performing-multiple-matrix-multiplications-in-cuda/23743838#23743838

void cublas_bmm_wrapper(hipblasHandle_t handle,
               torch::Tensor d_A, torch::Tensor d_B, torch::Tensor d_C,
               size_t a_rows, size_t a_cols, size_t b_cols, size_t b_rows,
               size_t batch_dim, bool transa, bool transb) {
    
    // ==============
    timestamp_t t0 = get_timestamp();
    float *d_A_arr = d_A.data_ptr<float>();
    float *d_B_arr = d_B.data_ptr<float>();
    float *d_C_arr = d_C.data_ptr<float>();
    /*
    printf("chkpt1\n");
    float *C;
    gpuErrchk(hipMalloc(&C, a_rows * b_cols * sizeof(float)));
    gpuErrchk(hipMemcpy(C, d_C_arr, a_rows * b_cols * sizeof(float), hipMemcpyHostToDevice));
    printf("chkpt2\n");
    dim3 threads_per_block(NUM_THREADS);
    dim3 C_blocks_per_grid((a_rows * b_cols + NUM_THREADS - 1)/NUM_THREADS);
    check_equal<<<threads_per_block, C_blocks_per_grid>>>(C, d_C_arr, a_rows, b_cols);
    printf("chkpt3\n");
    gpuErrchk(hipDeviceSynchronize());
    */
    timestamp_t t1 = get_timestamp();
    double secs = (t1 - t0) / 1000000.0L;
    printf("Preprocessing: %f\n", secs);

    t0 = get_timestamp();
    const float alpha = 1.0f, beta = 0.0f;

    hipblasOperation_t trans_a = (!transb) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    hipblasOperation_t trans_b = (!transa) ? HIPBLAS_OP_N : HIPBLAS_OP_T;

    hipblasStatus_t status = hipblasSgemmStridedBatched(handle, trans_a, trans_b
                                       , b_cols, a_rows, b_rows
                                       , &alpha, d_B_arr, b_cols, b_rows * b_cols
                                       , d_A_arr, b_rows, a_rows * b_rows
                                       , &beta, d_C_arr, b_cols, a_rows * b_cols
                                       , batch_dim);
    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        std::cerr << "Kernel execution error.";
    }
    /*
    gpuErrchk(hipDeviceSynchronize());
    printf("chkpt4\n");
    //gpuErrchk(hipMemcpy(C, d_C_arr, a_rows * b_cols * sizeof(float), hipMemcpyHostToDevice));
    //check_equal<<<threads_per_block, C_blocks_per_grid>>>(C, d_C_arr, a_rows, b_cols);
    //gpuErrchk(hipDeviceSynchronize());
    printf("chkpt5\n");
    gpuErrchk(hipMemcpy(C, d_C_arr, a_rows * b_cols * sizeof(float), hipMemcpyDeviceToDevice));
    check_equal<<<threads_per_block, C_blocks_per_grid>>>(C, d_C_arr, a_rows, b_cols);
    gpuErrchk(hipDeviceSynchronize());
    printf("chkpt6\n");
    */

    gpuErrchk(hipStreamSynchronize(0));
    t1 = get_timestamp();
    secs = (t1 - t0) / 1000000.0L;
    printf("Batch GEMM: %f\n", secs);
}

void cublas_4d_bmm_wrapper(hipblasHandle_t handle,
               torch::Tensor d_A, torch::Tensor d_B, torch::Tensor d_C,
               size_t a_rows, size_t a_cols, size_t b_cols, size_t b_rows,
               size_t batch_dim1, size_t batch_dim2,
               bool transa, bool transb) {

    float *d_A_arr = d_A.data_ptr<float>();
    float *d_B_arr = d_B.data_ptr<float>();
    float *d_C_arr = d_C.data_ptr<float>();
    const float alpha = 1.0f, beta = 0.0f;

    hipblasOperation_t trans_a = (!transa) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    hipblasOperation_t trans_b = (!transb) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    printf("transa: %d\n", (int) transa);
    printf("transb: %d\n", (int) transb);
    // (64, 512) (64, 256) transa
    // m = 512, n = 256, k = 64
    // m = a_cols -> a_cols
    // n = b_cols -> b_cols
    // k = b_rows -> b_rows 
    // (512, 64) (256, 64) transb
    // m = 512, n = 256, k = 64
    // m = a_rows -> a_rows
    // n = b_rows -> b_rows
    // k = b_cols -> b_cols
    // (512, 64) (64, 256)
    // m = 512, n = 256, k = 64
    // m = a_rows -> b_cols
    // n = b_cols -> a_rows
    // k = b_rows -> b_rows

    size_t m = 0;
    size_t n = 0;
    size_t k = 0;

    if (trans_a && trans_b) {
        m = a_rows;
        n = b_cols;
        k = b_rows;
    } else if (trans_a) {
        m = a_cols;
        n = b_cols;
        k = b_rows;
    } else if (trans_b) {
        m = a_rows;
        n = b_rows;
        k = b_cols;
    } else {
        m = b_cols;
        n = a_rows;
        k = b_rows;
    }

    if (trans_b) {
        m = a_rows;
        n = b_rows;
        k = b_cols;
        hipblasStatus_t status = hipblasSgemmStridedBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_T
                                       , m, n, k
                                       , &alpha, d_B_arr, n, b_rows * b_cols
                                       , d_A_arr, m, a_rows * a_cols
                                       , &beta, d_C_arr, a_rows, a_rows * b_rows
                                       , batch_dim1 * batch_dim2);

        if (status != HIPBLAS_STATUS_SUCCESS)
        {
            std::cerr << "Kernel execution error.";
        }
    } else {
        hipblasStatus_t status = hipblasSgemmStridedBatched(handle, trans_b, trans_a
                                       , m, n, k
                                       , &alpha, d_B_arr, m, b_rows * b_cols
                                       , d_A_arr, k, a_rows * b_rows
                                       , &beta, d_C_arr, m, a_rows * b_cols
                                       , batch_dim1 * batch_dim2);
    
        if (status != HIPBLAS_STATUS_SUCCESS)
        {
            std::cerr << "Kernel execution error.";
        }
    }

    
}

/*
cuSPARSE Kernels
https://stackoverflow.com/questions/29688627/sparse-matrix-matrix-multiplication-in-cuda-using-cusparse

Sparse (CSR) * Dense matmul
A * B = C

(m x k) * (k * n) = (m x n)
note: row_ind.len = lda + 1
*/
void cusparse_mm_wrapper(hipsparseHandle_t handle,
                         double *d_A, int *d_A_ColIndices, int *d_A_RowIndices,
                         int nnzA, int A_rowptr_size,
                         double *d_B_dense, int B_rows, int B_cols,
                         double *d_C_dense)
{
    // Initialize cuSPARSE
    // hipsparseHandle_t handle;
    // cusparseSafeCall(hipsparseCreate(&handle));
    const int m = A_rowptr_size - 1;
    const int k = B_rows;
    const int n = B_cols;

    // Descriptor for sparse matrix A
    hipsparseMatDescr_t descrA;
    cusparseSafeCall(hipsparseCreateMatDescr(&descrA));
    cusparseSafeCall(hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL));
    cusparseSafeCall(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ONE));
    // Descriptor for sparse matrix B
    hipsparseMatDescr_t descrB;
    cusparseSafeCall(hipsparseCreateMatDescr(&descrB));
    cusparseSafeCall(hipsparseSetMatType(descrB, HIPSPARSE_MATRIX_TYPE_GENERAL));
    cusparseSafeCall(hipsparseSetMatIndexBase(descrB, HIPSPARSE_INDEX_BASE_ONE));
    // Descriptor for sparse matrix C
    hipsparseMatDescr_t descrC;
    cusparseSafeCall(hipsparseCreateMatDescr(&descrC));
    cusparseSafeCall(hipsparseSetMatType(descrC, HIPSPARSE_MATRIX_TYPE_GENERAL));
    cusparseSafeCall(hipsparseSetMatIndexBase(descrC, HIPSPARSE_INDEX_BASE_ONE));

    int nnzB = 0; //   Number of nonzero elements in dense matrix B
    // Device side number of nonzero elements per row of matrix B
    int *d_nnzPerVectorB;
    gpuErrchk(hipMalloc(&d_nnzPerVectorB, k * sizeof(*d_nnzPerVectorB)));
    //cusparseSafeCall(hipsparseDnnz(handle, HIPSPARSE_DIRECTION_ROW, k, n, descrB, d_B_dense, k, d_nnzPerVectorB, &nnzB));

    // Device side sparse matrix B
    double *d_B;
    gpuErrchk(hipMalloc(&d_B, nnzB * sizeof(*d_B)));
    int *d_B_RowIndices;
    gpuErrchk(hipMalloc(&d_B_RowIndices, (k + 1) * sizeof(*d_B_RowIndices)));
    int *d_B_ColIndices;
    gpuErrchk(hipMalloc(&d_B_ColIndices, nnzB * sizeof(*d_B_ColIndices)));
    // Dense B to Sparse B
    //cusparseSafeCall(hipsparseDdense2csr(handle, k, n, descrB, d_B_dense, k, d_nnzPerVectorB, d_B, d_B_RowIndices, d_B_ColIndices));

    // Device side sparse matrix C
    int *d_C_RowIndices;
    gpuErrchk(hipMalloc(&d_C_RowIndices, (m + 1) * sizeof(*d_C_RowIndices)));

    // Performing the matrix - matrix multiplication
    int baseC, nnzC = 0;
    // nnzTotalDevHostPtr points to host memory
    int *nnzTotalDevHostPtr = &nnzC;

    //cusparseSafeCall(hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST));

    //cusparseSafeCall(hipsparseXcsrgemmNnz(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, n, k, descrA, nnzA,
    //                                     d_A_RowIndices, d_A_ColIndices, descrB, nnzB, d_B_RowIndices, d_B_ColIndices, descrC, d_C_RowIndices,
    //                                     nnzTotalDevHostPtr));
    if (nnzTotalDevHostPtr != NULL)
        nnzC = *nnzTotalDevHostPtr;
    else
    {
        hipMemcpy(&nnzC, d_C_RowIndices + m, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&baseC, d_C_RowIndices, sizeof(int), hipMemcpyDeviceToHost);
        nnzC -= baseC;
    }

    // device side sparse matrix C
    double *d_C;
    gpuErrchk(hipMalloc(&d_C, nnzC * sizeof(double)));
    int *d_C_ColIndices;
    gpuErrchk(hipMalloc(&d_C_ColIndices, nnzC * sizeof(int)));

    //cusparseSafeCall(hipsparseDcsrgemm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, n, k, descrA, nnzA,
    //                                  d_A, d_A_RowIndices, d_A_ColIndices, descrB, nnzB, d_B, d_B_RowIndices, d_B_ColIndices, descrC,
    //                                  d_C, d_C_RowIndices, d_C_ColIndices));

    //cusparseSafeCall(hipsparseDcsr2dense(handle, m, n, descrC, d_C, d_C_RowIndices, d_C_ColIndices, d_C_dense, m));

    hipFree(d_nnzPerVectorB);

    hipFree(d_B);
    hipFree(d_B_RowIndices);
    hipFree(d_B_ColIndices);

    hipFree(d_C);
    hipFree(d_C_RowIndices);
    hipFree(d_C_ColIndices);

    return;
}

void dense_to_csr(hipsparseHandle_t handle, 
                  double *d_A_dense, const int Nrows, const int Ncols,
                  double **d_A_val, int **d_A_colind, int **d_A_rowptr, int *nnzA)
{
    // Descriptor for sparse matrix A
    hipsparseMatDescr_t descrA;
    cusparseSafeCall(hipsparseCreateMatDescr(&descrA));
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);

    int nnz = 0;           //   Number of nonzero elements in dense matrix
    const int lda = Nrows; //   Leading dimension of dense matrix
    // Device side number of nonzero elements per row
    int *d_nnzPerVector;
    gpuErrchk(hipMalloc(&d_nnzPerVector, Nrows * sizeof(*d_nnzPerVector)));
    //cusparseSafeCall(hipsparseDnnz(handle, HIPSPARSE_DIRECTION_ROW, Nrows, Ncols, descrA, d_A_dense, lda, d_nnzPerVector, &nnz));

    // Device side sparse matrix
    double *d_A;
    gpuErrchk(hipMalloc(&d_A, nnz * sizeof(*d_A)));
    int *d_A_RowIndices;
    gpuErrchk(hipMalloc(&d_A_RowIndices, (Nrows + 1) * sizeof(*d_A_RowIndices)));
    int *d_A_ColIndices;
    gpuErrchk(hipMalloc(&d_A_ColIndices, nnz * sizeof(*d_A_ColIndices)));

    //cusparseSafeCall(hipsparseDdense2csr(handle, Nrows, Ncols, descrA, d_A_dense, lda, d_nnzPerVector, d_A, d_A_RowIndices, d_A_ColIndices));

    *d_A_val = d_A;
    *d_A_rowptr = d_A_RowIndices;
    *d_A_colind = d_A_ColIndices;
    *nnzA = nnz;

    gpuErrchk(hipFree(d_nnzPerVector));
    return;
}

int roundoff(int v, int d) {
    return (v + d - 1) / d * d;
}

void LtIgemmTensor(hipblasLtHandle_t ltHandle,
                   int m,
                   int n,
                   int k,
                   const float *A,
                   int lda,
                   const float *B,
                   int ldb,
                   float *C,
                   int ldc) {return;}
    /*
    hipblasLtMatmulDesc_t matmulDesc = NULL;
    hipblasLtMatrixLayout_t Adesc = NULL, Bdesc = NULL, Cdesc = NULL;
    int32_t alpha = 1, beta = 0;
    hipblasOperation_t opTranspose = HIPBLAS_OP_T;

    // tensor op igemm kernels require specialized memory order of data
    hipblasLtMatrixTransformDesc_t transformDesc = NULL;
    float *Atransform = NULL, *Btransform = NULL;
    float *Ctransform                   = NULL;
    hipblasLtMatrixLayout_t AtransformDesc = NULL, BtransformDesc = NULL, CtransformDesc = NULL;
    float transformAlpha = 1.0f, transformBeta = 0.0f;
    hipblasLtOrder_t order_COL32       = CUBLASLT_ORDER_COL32;
    hipblasLtOrder_t order_COL4_4R2_8C = CUBLASLT_ORDER_COL4_4R2_8C;

    int ldatransform = 32 * m;
    int ldbtransform = 32 * roundoff(n, 8);
    int ldctransform = 32 * m;

    checkCudaStatus(hipMalloc(reinterpret_cast<void**>(&Atransform), sizeof(float) * roundoff(k, 32) / 32 * ldatransform));
    checkCudaStatus(hipMalloc(reinterpret_cast<void**>(&Btransform), sizeof(float) * roundoff(k, 32) / 32 * ldbtransform));
    checkCudaStatus(hipMalloc(reinterpret_cast<void**>(&Ctransform), sizeof(float) * roundoff(n, 32) / 32 * ldctransform));

    checkCublasStatus(hipblasLtMatrixTransformDescCreate(&transformDesc, HIP_R_32F));

    //checkCublasStatus(hipblasLtMatmulDescCreate(&matmulDesc, HIP_R_32F));
    // tensor op igemm kernels only support NT gemm
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &opTranspose, sizeof(opTranspose)));

    // ---------------------------------------------------------------------------------------------
    // create descriptors for original matrices

    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_32F, m, k, lda));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_32F, k, n, ldb));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_32F, m, n, ldc));

    // ---------------------------------------------------------------------------------------------
    // create descriptors for transformed matrices

    checkCublasStatus(hipblasLtMatrixLayoutCreate(&AtransformDesc, HIP_R_32F, m, k, ldatransform));
    checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(AtransformDesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &order_COL32, sizeof(order_COL32)));

    // data memory order is set to CUBLASLT_ORDER_COL4_4R2_8C in order to achieve best performance on Turing devices.
    // for best performance on Ampere, consider setting the memory order to CUBLASLT_ORDER_COL32_2R_4R4.
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&BtransformDesc, HIP_R_32F, n, k, ldbtransform));
    checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(BtransformDesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &order_COL4_4R2_8C, sizeof(order_COL4_4R2_8C)));

    checkCublasStatus(hipblasLtMatrixLayoutCreate(&CtransformDesc, HIP_R_32F, m, n, ldctransform));
    checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(CtransformDesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &order_COL32, sizeof(order_COL32)));

    // ---------------------------------------------------------------------------------------------
    // transforms and computation

    checkCublasStatus(hipblasLtMatrixTransform(ltHandle, transformDesc, &transformAlpha, A, Adesc, &transformBeta, NULL, NULL, Atransform, AtransformDesc, 0));

    // B matrix is non-transposed, but transposed matrix is needed - add transpose operation in matrix transform.
    checkCublasStatus(hipblasLtMatrixTransformDescSetAttribute(transformDesc, HIPBLASLT_MATRIX_TRANSFORM_DESC_TRANSA, &opTranspose, sizeof(opTranspose)));

    checkCublasStatus(hipblasLtMatrixTransform(ltHandle, transformDesc, &transformAlpha, B, Bdesc, &transformBeta, NULL, NULL, Btransform, BtransformDesc, 0));

    // no need to transform C matrix as beta is assumed to be 0
    checkCublasStatus(hipblasLtMatmul(ltHandle,
                                     matmulDesc,
                                     &alpha,
                                     Atransform,
                                     AtransformDesc,
                                     Btransform,
                                     BtransformDesc,
                                     &beta,
                                     Ctransform,
                                     CtransformDesc,
                                     Ctransform,
                                     CtransformDesc,
                                     NULL,
                                     NULL,
                                     0,
                                     0));

    opTranspose = HIPBLAS_OP_N;
    checkCublasStatus(hipblasLtMatrixTransformDescSetAttribute(transformDesc, HIPBLASLT_MATRIX_TRANSFORM_DESC_TRANSA, &opTranspose, sizeof(opTranspose)));

    // transform outputs to COL order
    checkCublasStatus(hipblasLtMatrixTransform(ltHandle, transformDesc, &transformAlpha, Ctransform, CtransformDesc, &transformBeta, NULL, NULL, C, Cdesc, 0));

    // descriptors are no longer needed as all GPU work was already enqueued
    if (CtransformDesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(CtransformDesc));
    if (BtransformDesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(BtransformDesc));
    if (AtransformDesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(AtransformDesc));
    if (Cdesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Cdesc));
    if (Bdesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Bdesc));
    if (Adesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Adesc));
    if (matmulDesc) checkCublasStatus(hipblasLtMatmulDescDestroy(matmulDesc));
    if (transformDesc) checkCublasStatus(hipblasLtMatrixTransformDescDestroy(transformDesc));

    // wait until device is done before freeing transformed buffers
    checkCudaStatus(hipDeviceSynchronize());
    if (Ctransform) checkCudaStatus(hipFree(Ctransform));
    if (Btransform) checkCudaStatus(hipFree(Btransform));
    if (Atransform) checkCudaStatus(hipFree(Atransform));
}
*/
#endif // __MM_KERNEL_H__
