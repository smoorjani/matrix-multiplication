#include "hip/hip_runtime.h"
#ifndef __MM_KERNEL_H__
#define __MM_KERNEL_H__

#include <iostream>
#include <array>
#include <torch/extension.h>
#include "Utilities.cuh"

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsparse.h>
#include <hipblaslt.h>

#include <sys/time.h>
typedef unsigned long long timestamp_t;

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

static timestamp_t get_timestamp () {
    struct timeval now;
    gettimeofday (&now, NULL);
    return  now.tv_usec + (timestamp_t)now.tv_sec * 1000000;
}

#define NUM_THREADS (64)

__global__ void dummyKernel()
{
    int tid = threadIdx.x+blockIdx.x*blockDim.x;
    printf("%d\n", tid);
}

void dummy_kernel_launch() {
    dim3 threads_per_block(NUM_THREADS);
    dim3 blocks_per_grid(NUM_THREADS);
    dummyKernel<<<blocks_per_grid, threads_per_block>>>();
    checkCudaStatus(hipDeviceSynchronize());
}

__global__ void check_equal(float *d_Arr, float *h_Arr, size_t rows, size_t cols)
{
    int tid = threadIdx.x+blockIdx.x*blockDim.x;
    if (tid >= rows * cols) {
        return;
    }

    if (d_Arr[tid] == h_Arr[tid]) {
        printf("Equal %d\n", tid);
    } else {
        printf("Not Equal %d\n", tid);
    }
}

void cublas_mm_wrapper(hipblasHandle_t handle,
                       torch::Tensor d_A, torch::Tensor d_B, torch::Tensor d_C,
                       int a_rows, int a_cols, int b_rows, int b_cols,
                       bool transa, bool transb) {

    float *d_A_arr = d_A.data_ptr<float>();
    float *d_B_arr = d_B.data_ptr<float>();
    float *d_C_arr = d_C.data_ptr<float>();

    hipblasOperation_t trans_a = (!transb) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    hipblasOperation_t trans_b = (!transa) ? HIPBLAS_OP_N : HIPBLAS_OP_T;

    int m = b_cols;
    int n = a_rows;
    int k = b_rows;
    int ldb = b_cols;
    int lda = b_rows;
    int ldc = b_cols;

    if (transb && transa) {
        m = b_rows;
        n = a_cols;
        k = b_cols;
        lda = n;
        ldb = k;
	    ldc = m;
    } else if (transa) {
        m = b_cols;
        n = a_cols;
        k = b_rows;
        lda = n;
        ldb = m;
	    ldc = m;
    } else if (transb) {
        m = b_rows;
        n = a_rows;
        k = b_cols;
        lda = k;
        ldb = k;
	    ldc = m;
    }

    float alpha = 1.0;
    float beta = 0.0;
    checkCublasStatus(hipblasSgemm(
        handle, trans_a, trans_b,
        m, n, k, &alpha,
        d_B_arr, ldb,
        d_A_arr, lda, &beta,
        d_C_arr, ldc));
}


// https://stackoverflow.com/questions/23743384/how-performing-multiple-matrix-multiplications-in-cuda/23743838#23743838
void cublas_bmm_wrapper(hipblasHandle_t handle,
               torch::Tensor d_A, torch::Tensor d_B, torch::Tensor d_C,
               size_t a_rows, size_t a_cols, size_t b_cols, size_t b_rows,
               size_t batch_dim, bool transa, bool transb) {
    float *d_A_arr = d_A.data_ptr<float>();
    float *d_B_arr = d_B.data_ptr<float>();
    float *d_C_arr = d_C.data_ptr<float>();

    const float alpha = 1.0f, beta = 0.0f;

    hipblasOperation_t trans_a = (!transb) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    hipblasOperation_t trans_b = (!transa) ? HIPBLAS_OP_N : HIPBLAS_OP_T;

    int m = b_cols;
    int n = a_rows;
    int k = b_rows;

    int ldb = b_cols;
    int lda = b_rows;
    int ldc = b_cols;

    if (transb && transa) {
        m = b_rows;
        n = a_cols;
        k = b_cols;
        lda = n;
        ldb = k;
        ldc = m;
    } else if (transa) {
        m = b_cols;
        n = a_cols;
        k = b_rows;
        lda = n;
        ldb = m;
	    ldc = m;
    } else if (transb) {
        m = b_rows;
        n = a_rows;
        k = b_cols;
        lda = k;
        ldb = k;
	    ldc = m;
    }

    checkCublasStatus(hipblasSgemmStridedBatched(handle, trans_a, trans_b
                                       , m, n, k
                                       , &alpha, d_B_arr, ldb, m * k
                                       , d_A_arr, lda, n * k
                                       , &beta, d_C_arr, ldc, m * n
                                       , batch_dim));
}

void cublas_4d_bmm_wrapper(hipblasHandle_t handle,
               torch::Tensor d_A, torch::Tensor d_B, torch::Tensor d_C,
               size_t a_rows, size_t a_cols, size_t b_cols, size_t b_rows,
               size_t batch_dim1, size_t batch_dim2,
               bool transa, bool transb) {

    float *d_A_arr = d_A.data_ptr<float>();
    float *d_B_arr = d_B.data_ptr<float>();
    float *d_C_arr = d_C.data_ptr<float>();

    hipblasOperation_t trans_a = (!transb) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    hipblasOperation_t trans_b = (!transa) ? HIPBLAS_OP_N : HIPBLAS_OP_T;

    int m = b_cols;
    int n = a_rows;
    int k = b_rows;

    int ldb = b_cols;
    int lda = b_rows;
    int ldc = b_cols;

    if (transb && transa) {
        m = b_rows;
        n = a_cols;
        k = b_cols;
        lda = n;
        ldb = k;
        ldc = m;
    } else if (transa) {
        m = b_cols;
        n = a_cols;
        k = b_rows;
        lda = n;
        ldb = m;
	ldc = m;
    } else if (transb) {
        m = b_rows;
        n = a_rows;
        k = b_cols;
        lda = k;
        ldb = k;
	ldc = m;
    }

    const float alpha = 1.0f, beta = 0.0f;
    checkCublasStatus(hipblasSgemmStridedBatched(handle, trans_a, trans_b
                                       , m, n, k
                                       , &alpha, d_B_arr, ldb, m * k
                                       , d_A_arr, lda, n * k
                                       , &beta, d_C_arr, ldc, m * n
                                       , batch_dim1 * batch_dim2));
}

/*
cuSPARSE Kernels
https://stackoverflow.com/questions/29688627/sparse-matrix-matrix-multiplication-in-cuda-using-cusparse

Sparse (CSR) * Dense matmul
A * B = C

(m x k) * (k * n) = (m x n)
note: row_ind.len = lda + 1
*/
void cusparse_mm_wrapper(hipsparseHandle_t handle,
                         float *dA_values, int *dA_columns, int *dA_csrOffsets,
                         int A_nnz, int A_num_rows, int A_num_cols,
                         torch::Tensor B, int B_num_rows, int B_num_cols,
                         torch::Tensor C)
{
    float *dB = B.data_ptr<float>();
    float *dC = C.data_ptr<float>();

    int ldb = B_num_rows;
    int ldc = A_num_rows;

    // CUSPARSE APIs
    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t matB, matC;
    void* dBuffer = NULL;
    size_t bufferSize = 0;
    // Create sparse matrix A in CSR format
    CHECK_CUSPARSE(hipsparseCreateCsr(&matA, A_num_rows, A_num_cols, A_nnz,
                                      dA_csrOffsets, dA_columns, dA_values,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F))
    // Create dense matrix B
    CHECK_CUSPARSE(hipsparseCreateDnMat(&matB, A_num_cols, B_num_cols, ldb, dB,
                                        HIP_R_32F, HIPSPARSE_ORDER_COL))
    // Create dense matrix C
    CHECK_CUSPARSE(hipsparseCreateDnMat(&matC, A_num_rows, B_num_cols, ldc, dC,
                                        HIP_R_32F, HIPSPARSE_ORDER_COL))
    // allocate an external buffer if needed
    float alpha = 1.0f;
    float beta = 0.0f;
    CHECK_CUSPARSE(hipsparseSpMM_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           &alpha, matA, matB, &beta, matC, HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize))
    CHECK_CUDA(hipMalloc(&dBuffer, bufferSize))

    // execute SpMM
    CHECK_CUSPARSE(hipsparseSpMM(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                &alpha, matA, matB, &beta, matC, HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer))

    // destroy matrix/vector descriptors
    CHECK_CUSPARSE(hipsparseDestroySpMat(matA))
    CHECK_CUSPARSE(hipsparseDestroyDnMat(matB))
    CHECK_CUSPARSE(hipsparseDestroyDnMat(matC))

    // device memory deallocation
    // CHECK_CUDA(hipFree(dBuffer))
    // CHECK_CUDA(hipFree(dA_csrOffsets))
    // CHECK_CUDA(hipFree(dA_columns))
    // CHECK_CUDA(hipFree(dA_values))
    // CHECK_CUDA(hipFree(dB))
    CHECK_CUDA( hipFree(dC) )
    return;
}

void dense_to_csr(hipsparseHandle_t handle, 
                  torch::Tensor dense, const int num_rows, const int num_cols,
                  float *d_csr_values, int *d_csr_columns, int *d_csr_offsets, int *nnz)
{
    int ld = num_cols;
    float *d_dense = dense.data_ptr<float>();
                           
    // CUSPARSE APIs
    hipsparseSpMatDescr_t matB;
    hipsparseDnMatDescr_t matA;
    void* dBuffer = NULL;
    size_t bufferSize = 0;

    // Allocate memory for offsets
    CHECK_CUDA(hipMalloc((void**) &d_csr_offsets, (num_rows + 1) * sizeof(int)))

    // Create dense matrix A
    CHECK_CUSPARSE(hipsparseCreateDnMat(&matA, num_rows, num_cols, ld, d_dense, HIP_R_32F, HIPSPARSE_ORDER_ROW))
    // Create sparse matrix B in CSR format
    CHECK_CUSPARSE(hipsparseCreateCsr(&matB, num_rows, num_cols, 0, d_csr_offsets, NULL, NULL,
                                     HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F))

    // allocate an external buffer if needed
    CHECK_CUSPARSE(hipsparseDenseToSparse_bufferSize(handle, matA, matB, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, &bufferSize))
    CHECK_CUDA(hipMalloc(&dBuffer, bufferSize))

    // execute Sparse to Dense conversion
    CHECK_CUSPARSE(hipsparseDenseToSparse_analysis(handle, matA, matB, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, dBuffer))

    // get number of non-zero elements
    int64_t num_rows_tmp, num_cols_tmp, nnz_tmp;
    CHECK_CUSPARSE(hipsparseSpMatGetSize(matB, &num_rows_tmp, &num_cols_tmp, &nnz_tmp))
    *nnz = nnz_tmp;

    // allocate CSR column indices and values
    CHECK_CUDA(hipMalloc((void**) &d_csr_columns, nnz_tmp * sizeof(int)))
    CHECK_CUDA(hipMalloc((void**) &d_csr_values,  nnz_tmp * sizeof(float)))
    // reset offsets, column indices, and values pointers
    CHECK_CUSPARSE(hipsparseCsrSetPointers(matB, d_csr_offsets, d_csr_columns, d_csr_values))
    // execute Sparse to Dense conversion
    CHECK_CUSPARSE(hipsparseDenseToSparse_convert(handle, matA, matB, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, dBuffer))
    // destroy matrix/vector descriptors
    CHECK_CUSPARSE(hipsparseDestroyDnMat(matA))
    CHECK_CUSPARSE(hipsparseDestroySpMat(matB))
    // device memory deallocation
    CHECK_CUDA(hipFree(dBuffer))
}




int roundoff(int v, int d) {
    return (v + d - 1) / d * d;
}

void LtIgemmTensor(hipblasLtHandle_t ltHandle,
                   int m,
                   int n,
                   int k,
                   const float *A,
                   int lda,
                   const float *B,
                   int ldb,
                   float *C,
                   int ldc) {return;}
    /*
    hipblasLtMatmulDesc_t matmulDesc = NULL;
    hipblasLtMatrixLayout_t Adesc = NULL, Bdesc = NULL, Cdesc = NULL;
    int32_t alpha = 1, beta = 0;
    hipblasOperation_t opTranspose = HIPBLAS_OP_T;

    // tensor op igemm kernels require specialized memory order of data
    hipblasLtMatrixTransformDesc_t transformDesc = NULL;
    float *Atransform = NULL, *Btransform = NULL;
    float *Ctransform                   = NULL;
    hipblasLtMatrixLayout_t AtransformDesc = NULL, BtransformDesc = NULL, CtransformDesc = NULL;
    float transformAlpha = 1.0f, transformBeta = 0.0f;
    hipblasLtOrder_t order_COL32       = CUBLASLT_ORDER_COL32;
    hipblasLtOrder_t order_COL4_4R2_8C = CUBLASLT_ORDER_COL4_4R2_8C;

    int ldatransform = 32 * m;
    int ldbtransform = 32 * roundoff(n, 8);
    int ldctransform = 32 * m;

    checkCudaStatus(hipMalloc(reinterpret_cast<void**>(&Atransform), sizeof(float) * roundoff(k, 32) / 32 * ldatransform));
    checkCudaStatus(hipMalloc(reinterpret_cast<void**>(&Btransform), sizeof(float) * roundoff(k, 32) / 32 * ldbtransform));
    checkCudaStatus(hipMalloc(reinterpret_cast<void**>(&Ctransform), sizeof(float) * roundoff(n, 32) / 32 * ldctransform));

    checkCublasStatus(hipblasLtMatrixTransformDescCreate(&transformDesc, HIP_R_32F));

    //checkCublasStatus(hipblasLtMatmulDescCreate(&matmulDesc, HIP_R_32F));
    // tensor op igemm kernels only support NT gemm
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &opTranspose, sizeof(opTranspose)));

    // ---------------------------------------------------------------------------------------------
    // create descriptors for original matrices

    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_32F, m, k, lda));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_32F, k, n, ldb));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_32F, m, n, ldc));

    // ---------------------------------------------------------------------------------------------
    // create descriptors for transformed matrices

    checkCublasStatus(hipblasLtMatrixLayoutCreate(&AtransformDesc, HIP_R_32F, m, k, ldatransform));
    checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(AtransformDesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &order_COL32, sizeof(order_COL32)));

    // data memory order is set to CUBLASLT_ORDER_COL4_4R2_8C in order to achieve best performance on Turing devices.
    // for best performance on Ampere, consider setting the memory order to CUBLASLT_ORDER_COL32_2R_4R4.
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&BtransformDesc, HIP_R_32F, n, k, ldbtransform));
    checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(BtransformDesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &order_COL4_4R2_8C, sizeof(order_COL4_4R2_8C)));

    checkCublasStatus(hipblasLtMatrixLayoutCreate(&CtransformDesc, HIP_R_32F, m, n, ldctransform));
    checkCublasStatus(hipblasLtMatrixLayoutSetAttribute(CtransformDesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &order_COL32, sizeof(order_COL32)));

    // ---------------------------------------------------------------------------------------------
    // transforms and computation

    checkCublasStatus(hipblasLtMatrixTransform(ltHandle, transformDesc, &transformAlpha, A, Adesc, &transformBeta, NULL, NULL, Atransform, AtransformDesc, 0));

    // B matrix is non-transposed, but transposed matrix is needed - add transpose operation in matrix transform.
    checkCublasStatus(hipblasLtMatrixTransformDescSetAttribute(transformDesc, HIPBLASLT_MATRIX_TRANSFORM_DESC_TRANSA, &opTranspose, sizeof(opTranspose)));

    checkCublasStatus(hipblasLtMatrixTransform(ltHandle, transformDesc, &transformAlpha, B, Bdesc, &transformBeta, NULL, NULL, Btransform, BtransformDesc, 0));

    // no need to transform C matrix as beta is assumed to be 0
    checkCublasStatus(hipblasLtMatmul(ltHandle,
                                     matmulDesc,
                                     &alpha,
                                     Atransform,
                                     AtransformDesc,
                                     Btransform,
                                     BtransformDesc,
                                     &beta,
                                     Ctransform,
                                     CtransformDesc,
                                     Ctransform,
                                     CtransformDesc,
                                     NULL,
                                     NULL,
                                     0,
                                     0));

    opTranspose = HIPBLAS_OP_N;
    checkCublasStatus(hipblasLtMatrixTransformDescSetAttribute(transformDesc, HIPBLASLT_MATRIX_TRANSFORM_DESC_TRANSA, &opTranspose, sizeof(opTranspose)));

    // transform outputs to COL order
    checkCublasStatus(hipblasLtMatrixTransform(ltHandle, transformDesc, &transformAlpha, Ctransform, CtransformDesc, &transformBeta, NULL, NULL, C, Cdesc, 0));

    // descriptors are no longer needed as all GPU work was already enqueued
    if (CtransformDesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(CtransformDesc));
    if (BtransformDesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(BtransformDesc));
    if (AtransformDesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(AtransformDesc));
    if (Cdesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Cdesc));
    if (Bdesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Bdesc));
    if (Adesc) checkCublasStatus(hipblasLtMatrixLayoutDestroy(Adesc));
    if (matmulDesc) checkCublasStatus(hipblasLtMatmulDescDestroy(matmulDesc));
    if (transformDesc) checkCublasStatus(hipblasLtMatrixTransformDescDestroy(transformDesc));

    // wait until device is done before freeing transformed buffers
    checkCudaStatus(hipDeviceSynchronize());
    if (Ctransform) checkCudaStatus(hipFree(Ctransform));
    if (Btransform) checkCudaStatus(hipFree(Btransform));
    if (Atransform) checkCudaStatus(hipFree(Atransform));
}
*/
#endif // __MM_KERNEL_H__
