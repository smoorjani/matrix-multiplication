#include "hip/hip_runtime.h"
#ifndef __MM_KERNEL_H__
#define __MM_KERNEL_H__

#include <iostream>
#include <torch/extension.h>
#include "Utilities.cuh"

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsparse.h>

//#define NUM_THREADS 64;
int NUM_THREADS = 64;

void cublas_mm_wrapper(hipblasHandle_t handle,
                       float *d_A, float *d_B, float *d_C,
                       int m, int k, int n) {
    hipMemset(d_C, 0, m * n * sizeof(float));
 
    float alpha = 1.0;
    float beta = 0.0;
    hipblasStatus_t status = hipblasSgemm(
        handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
        n, m, k, &alpha,
        d_B, n,
        d_A, k, &beta,
        d_C, n);

    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        std::cerr << "Kernel execution error.";
    }

}

__global__ void packed_accessor_kernel(
    // figure out if this is coalesced
    torch::PackedTensorAccessor32<float, 3> accessor,
    float** trace
) {
  int NUM_THREADS = 64;
  int batch_size = accessor.size(0);
  int n_rows = accessor.size(1);
  int n_cols = accessor.size(2);
    
  int idx_per_row = (int) ((float) (n_cols + NUM_THREADS - 1) / NUM_THREADS);
  // find thread id, row = threadid/64, col = threadid%64
  int threadId = threadIdx.x + blockDim.x * blockIdx.x;
  int batch = 0, row = 0, col = 0;

  if (n_cols >= NUM_THREADS) {
    int idx = threadId / NUM_THREADS;
    int off = threadId % NUM_THREADS;

    batch = idx / (idx_per_row * n_rows);
    row = (idx / idx_per_row) % n_rows;
    col = (idx % idx_per_row) * NUM_THREADS + off;
  } else {
    batch = threadId / (n_rows * n_cols);
    row = (threadId % (n_rows * n_cols))/ n_cols;
    col = threadId % n_cols;
  }

  if (batch >= batch_size || row >= n_rows || col >= n_cols) {
    return;
  }
  
  trace[batch][row * n_cols + col] = accessor[batch][row][col];
}


__global__ void packed_setter_kernel(
    torch::PackedTensorAccessor32<float, 3> accessor,
    float** trace
) {
  int NUM_THREADS = 64;
  int batch_size = accessor.size(0);
  int n_rows = accessor.size(1);
  int n_cols = accessor.size(2);
  int idx_per_row = (int) ((float) (n_cols + NUM_THREADS - 1) / NUM_THREADS);
  // find thread id, row = threadid/64, col = threadid%64
  int threadId = threadIdx.x + blockDim.x * blockIdx.x;
  int batch= 0, row = 0, col = 0;

  if (n_cols >= NUM_THREADS) {
    int idx = threadId / NUM_THREADS;
    int off = threadId % NUM_THREADS;

    batch = idx / (idx_per_row * n_rows);
    row = (idx / idx_per_row) % n_rows;
    col = (idx % idx_per_row) * NUM_THREADS + off;
  } else {
    batch = threadId / (n_rows * n_cols);
    row = (threadId % (n_rows * n_cols))/ n_cols;
    col = threadId % n_cols;
  }

  if (batch >= batch_size || row >= n_rows || col >= n_cols) {
    return;
  }

  accessor[batch][row][col] = trace[batch][row * n_cols + col];
}

// https://stackoverflow.com/questions/23743384/how-performing-multiple-matrix-multiplications-in-cuda/23743838#23743838

void cublas_bmm_wrapper_accessor(hipblasHandle_t handle,
               torch::Tensor d_A, torch::Tensor d_B, torch::Tensor d_C,
               size_t a_rows, size_t b_cols, size_t b_rows,
               size_t batch_dim) {

    //auto A_accessory = d_A.packed_accessor32<float,3>();
    //float **d_A_arry = NULL;
    //test_kernel<<<4,2>>>();
    //packed_accessor_kernel<<<(a_rows * b_rows + NUM_THREADS)/NUM_THREADS, NUM_THREADS>>>(A_accessory, d_A_arry);
    //hipDeviceSynchronize();
    //return;

    size_t a_size = sizeof(float) * a_rows * b_rows;
    size_t b_size = sizeof(float) * b_rows * b_cols;
    size_t c_size = sizeof(float) * a_rows * b_cols;

    float **d_A_arr, **d_B_arr, **d_C_arr;
    float **h_A_arr = (float **) malloc(batch_dim * sizeof(float*));
    float **h_B_arr = (float **) malloc(batch_dim * sizeof(float*));
    float **h_C_arr = (float **) malloc(batch_dim * sizeof(float*));

    gpuErrchk(hipMalloc((void **)&d_A_arr, batch_dim * sizeof(float *)));
    gpuErrchk(hipMalloc((void **)&d_B_arr, batch_dim * sizeof(float *)));
    gpuErrchk(hipMalloc((void **)&d_C_arr, batch_dim * sizeof(float *)));

    for (int i = 0; i < batch_dim; i++) {
        gpuErrchk(hipMalloc(&h_A_arr[i], a_size));
        gpuErrchk(hipMalloc(&h_B_arr[i], b_size));
        gpuErrchk(hipMalloc(&h_C_arr[i], c_size));
    }

    gpuErrchk(hipMemcpy(d_A_arr, h_A_arr, batch_dim * sizeof(float*), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_B_arr, h_B_arr, batch_dim * sizeof(float*), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_C_arr, h_C_arr, batch_dim * sizeof(float*), hipMemcpyHostToDevice));

    auto A_accessor = d_A.packed_accessor32<float,3>();
    auto B_accessor = d_B.packed_accessor32<float,3>();
    auto C_accessor = d_C.packed_accessor32<float,3>();

    // execute 1 time with a_rows threads
    dim3 thread_per_block(NUM_THREADS);
    dim3 A_blocks_per_grid((batch_dim * a_rows * b_rows + NUM_THREADS - 1)/NUM_THREADS);
    dim3 B_blocks_per_grid((batch_dim * b_rows * b_cols + NUM_THREADS - 1)/NUM_THREADS);
    dim3 C_blocks_per_grid((batch_dim * a_rows * b_cols + NUM_THREADS - 1)/NUM_THREADS);

    // <<<total threads/64 ,64>>>>
    packed_accessor_kernel<<<A_blocks_per_grid, thread_per_block>>>(A_accessor, d_A_arr);
    hipDeviceSynchronize();
    packed_accessor_kernel<<<B_blocks_per_grid, thread_per_block>>>(B_accessor, d_B_arr);
    hipDeviceSynchronize();
  
    const float alpha = 1.0f, beta = 0.0f;
       
    hipblasStatus_t status = hipblasSgemmBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N
                                       , b_cols, a_rows, b_rows
                                       , &alpha, d_B_arr, b_cols, d_A_arr, b_rows
                                       , &beta, d_C_arr, b_cols
                                       , batch_dim);
    
    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        std::cerr << "Kernel execution error.";
    }

    packed_setter_kernel<<<C_blocks_per_grid, thread_per_block>>>(C_accessor, d_C_arr);
    hipDeviceSynchronize();
    
    for (int i = 0; i < batch_dim; i++) {
        gpuErrchk(hipFree(h_A_arr[i]));
        gpuErrchk(hipFree(h_B_arr[i]));
        gpuErrchk(hipFree(h_C_arr[i]));
    }
    gpuErrchk(hipFree(d_A_arr));
    gpuErrchk(hipFree(d_B_arr));
    gpuErrchk(hipFree(d_C_arr));
    
    free(h_A_arr);
    free(h_B_arr);
    free(h_C_arr);
}


void cublas_bmm_wrapper(hipblasHandle_t handle,
               float *d_A, float *d_B, float *d_C,
               size_t a_rows, size_t b_cols, size_t b_rows,
               size_t batch_dim) {


    size_t c_size = sizeof(float) * a_rows * b_cols;
    size_t a_size = sizeof(float) * a_rows * b_rows;
    size_t b_size = sizeof(float) * b_rows * b_cols;

    float **d_A_arr, **d_B_arr;
    float **d_C_arr;

    gpuErrchk(hipMalloc((void **)&d_A_arr, batch_dim * sizeof(float *)));
    gpuErrchk(hipMalloc((void **)&d_B_arr, batch_dim * sizeof(float *)));
    gpuErrchk(hipMalloc((void **)&d_C_arr, batch_dim * sizeof(float *)));

    // assumes contiguous memory
    for (int i = 0; i < batch_dim; i++) {
        gpuErrchk(hipMemcpy(d_A_arr[i], d_A + (batch_dim * a_size * i), (batch_dim * a_size), hipMemcpyDeviceToDevice));
        gpuErrchk(hipMemcpy(d_B_arr[i], d_B + (batch_dim * b_size * i), (batch_dim * b_size), hipMemcpyDeviceToDevice));
        gpuErrchk(hipMemcpy(d_C_arr[i], d_C + (batch_dim * c_size * i), (batch_dim * c_size), hipMemcpyDeviceToDevice));
    }

    const float alpha = 1.0f, beta = 0.0f;
    hipblasStatus_t cublas_result = hipblasSgemmBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N
                                       , b_cols, a_rows, b_rows
                                       , &alpha, d_B_arr, b_cols, d_A_arr, b_rows
                                       , &beta, d_C_arr, b_cols
                                       , batch_dim);
    assert(cublas_result == HIPBLAS_STATUS_SUCCESS);

    gpuErrchk(hipFree(d_A_arr));
    gpuErrchk(hipFree(d_B_arr));
    gpuErrchk(hipFree(d_C_arr));
}


/*
cuSPARSE Kernels
https://stackoverflow.com/questions/29688627/sparse-matrix-matrix-multiplication-in-cuda-using-cusparse

Sparse (CSR) * Dense matmul
A * B = C

(m x k) * (k * n) = (m x n)
note: row_ind.len = lda + 1
*/
void cusparse_mm_wrapper(hipsparseHandle_t handle,
                         double *d_A, int *d_A_ColIndices, int *d_A_RowIndices,
                         int nnzA, int A_rowptr_size,
                         double *d_B_dense, int B_rows, int B_cols,
                         double *d_C_dense)
{
    // Initialize cuSPARSE
    // hipsparseHandle_t handle;
    // cusparseSafeCall(hipsparseCreate(&handle));
    const int m = A_rowptr_size - 1;
    const int k = B_rows;
    const int n = B_cols;

    // Descriptor for sparse matrix A
    hipsparseMatDescr_t descrA;
    cusparseSafeCall(hipsparseCreateMatDescr(&descrA));
    cusparseSafeCall(hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL));
    cusparseSafeCall(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ONE));
    // Descriptor for sparse matrix B
    hipsparseMatDescr_t descrB;
    cusparseSafeCall(hipsparseCreateMatDescr(&descrB));
    cusparseSafeCall(hipsparseSetMatType(descrB, HIPSPARSE_MATRIX_TYPE_GENERAL));
    cusparseSafeCall(hipsparseSetMatIndexBase(descrB, HIPSPARSE_INDEX_BASE_ONE));
    // Descriptor for sparse matrix C
    hipsparseMatDescr_t descrC;
    cusparseSafeCall(hipsparseCreateMatDescr(&descrC));
    cusparseSafeCall(hipsparseSetMatType(descrC, HIPSPARSE_MATRIX_TYPE_GENERAL));
    cusparseSafeCall(hipsparseSetMatIndexBase(descrC, HIPSPARSE_INDEX_BASE_ONE));

    int nnzB = 0; //   Number of nonzero elements in dense matrix B
    // Device side number of nonzero elements per row of matrix B
    int *d_nnzPerVectorB;
    gpuErrchk(hipMalloc(&d_nnzPerVectorB, k * sizeof(*d_nnzPerVectorB)));
    cusparseSafeCall(hipsparseDnnz(handle, HIPSPARSE_DIRECTION_ROW, k, n, descrB, d_B_dense, k, d_nnzPerVectorB, &nnzB));

    // Device side sparse matrix B
    double *d_B;
    gpuErrchk(hipMalloc(&d_B, nnzB * sizeof(*d_B)));
    int *d_B_RowIndices;
    gpuErrchk(hipMalloc(&d_B_RowIndices, (k + 1) * sizeof(*d_B_RowIndices)));
    int *d_B_ColIndices;
    gpuErrchk(hipMalloc(&d_B_ColIndices, nnzB * sizeof(*d_B_ColIndices)));
    // Dense B to Sparse B
    cusparseSafeCall(hipsparseDdense2csr(handle, k, n, descrB, d_B_dense, k, d_nnzPerVectorB, d_B, d_B_RowIndices, d_B_ColIndices));

    // Device side sparse matrix C
    int *d_C_RowIndices;
    gpuErrchk(hipMalloc(&d_C_RowIndices, (m + 1) * sizeof(*d_C_RowIndices)));

    // Performing the matrix - matrix multiplication
    int baseC, nnzC = 0;
    // nnzTotalDevHostPtr points to host memory
    int *nnzTotalDevHostPtr = &nnzC;

    cusparseSafeCall(hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST));

    cusparseSafeCall(hipsparseXcsrgemmNnz(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, n, k, descrA, nnzA,
                                         d_A_RowIndices, d_A_ColIndices, descrB, nnzB, d_B_RowIndices, d_B_ColIndices, descrC, d_C_RowIndices,
                                         nnzTotalDevHostPtr));
    if (nnzTotalDevHostPtr != NULL)
        nnzC = *nnzTotalDevHostPtr;
    else
    {
        hipMemcpy(&nnzC, d_C_RowIndices + m, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&baseC, d_C_RowIndices, sizeof(int), hipMemcpyDeviceToHost);
        nnzC -= baseC;
    }

    // device side sparse matrix C
    double *d_C;
    gpuErrchk(hipMalloc(&d_C, nnzC * sizeof(double)));
    int *d_C_ColIndices;
    gpuErrchk(hipMalloc(&d_C_ColIndices, nnzC * sizeof(int)));

    cusparseSafeCall(hipsparseDcsrgemm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, n, k, descrA, nnzA,
                                      d_A, d_A_RowIndices, d_A_ColIndices, descrB, nnzB, d_B, d_B_RowIndices, d_B_ColIndices, descrC,
                                      d_C, d_C_RowIndices, d_C_ColIndices));

    cusparseSafeCall(hipsparseDcsr2dense(handle, m, n, descrC, d_C, d_C_RowIndices, d_C_ColIndices, d_C_dense, m));

    hipFree(d_nnzPerVectorB);

    hipFree(d_B);
    hipFree(d_B_RowIndices);
    hipFree(d_B_ColIndices);

    hipFree(d_C);
    hipFree(d_C_RowIndices);
    hipFree(d_C_ColIndices);

    return;
}

void dense_to_csr(hipsparseHandle_t handle, 
                  double *d_A_dense, const int Nrows, const int Ncols,
                  double **d_A_val, int **d_A_colind, int **d_A_rowptr, int *nnzA)
{
    // Descriptor for sparse matrix A
    hipsparseMatDescr_t descrA;
    cusparseSafeCall(hipsparseCreateMatDescr(&descrA));
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);

    int nnz = 0;           //   Number of nonzero elements in dense matrix
    const int lda = Nrows; //   Leading dimension of dense matrix
    // Device side number of nonzero elements per row
    int *d_nnzPerVector;
    gpuErrchk(hipMalloc(&d_nnzPerVector, Nrows * sizeof(*d_nnzPerVector)));
    cusparseSafeCall(hipsparseDnnz(handle, HIPSPARSE_DIRECTION_ROW, Nrows, Ncols, descrA, d_A_dense, lda, d_nnzPerVector, &nnz));

    // Device side sparse matrix
    double *d_A;
    gpuErrchk(hipMalloc(&d_A, nnz * sizeof(*d_A)));
    int *d_A_RowIndices;
    gpuErrchk(hipMalloc(&d_A_RowIndices, (Nrows + 1) * sizeof(*d_A_RowIndices)));
    int *d_A_ColIndices;
    gpuErrchk(hipMalloc(&d_A_ColIndices, nnz * sizeof(*d_A_ColIndices)));

    cusparseSafeCall(hipsparseDdense2csr(handle, Nrows, Ncols, descrA, d_A_dense, lda, d_nnzPerVector, d_A, d_A_RowIndices, d_A_ColIndices));

    *d_A_val = d_A;
    *d_A_rowptr = d_A_RowIndices;
    *d_A_colind = d_A_ColIndices;
    *nnzA = nnz;

    gpuErrchk(hipFree(d_nnzPerVector));
    return;
}

#endif // __MM_KERNEL_H__
