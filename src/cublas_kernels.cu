#include "hip/hip_runtime.h"
#ifndef __CUBLAS_BMM_KERNEL_H__
#define __CUBLAS_BMM_KERNEL_H__

// https://stackoverflow.com/questions/23743384/how-performing-multiple-matrix-multiplications-in-cuda/23743838#23743838

#include <iostream>
#include <torch/extension.h>
#include "Utilities.cuh"

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsparse.h>
/*
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
*/


void cublas_mm_wrapper(hipblasHandle_t handle,
                       float *d_A, float *d_B, float *d_C,
                       int m, int k, int n) {
    hipMemset(d_C, 0, m * n * sizeof(float));
 

    /* 
    printf("m: %d, k: %d, n: %d\n", m, k, n);
 
    float *h_A = (float *) malloc(sizeof(float) * m * k);
    float *h_B = (float *) malloc(sizeof(float) * k * n);
    gpuErrchk(hipMemcpy(h_A, d_A, sizeof(float) * m * k, hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_B, d_B, sizeof(float) * k * n, hipMemcpyDeviceToHost));

    print_arr(h_A, m, k);
    print_arr(h_B, k, n);
    */

    float alpha = 1.0;
    float beta = 0.0;
    hipblasStatus_t status = hipblasSgemm(
        handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
        n, m, k, &alpha,
        d_B, n,
        d_A, k, &beta,
        d_C, n);

    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        std::cerr << "Kernel execution error.";
    }

}


void printArrayS(float *ptr, int rows, int cols, char mode, char *name) {
    printf("%s\n", name);
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            if (mode == 'B') /* Normal mode */ {
                if (ptr[i * cols + j] >= 0)
                    printf(" %3.6f ", ptr[i * cols + j]);
                else
                    printf("%3.6f ", ptr[i * cols + j]);
            } else /* Transpose mode */ {
                if (ptr[j * rows + i] >= 0)
                    printf("%3.6f ", ptr[j * rows + i]);
                else
                    printf("%3.6f ", ptr[j * rows + i]);
            }
        }
        printf("\n");
    }
}

/*
__global__ void packed_accessor_kernel(
    torch::PackedTensorAccessor32<float, 3> accessor,
    float** trace) {
  int i=threadIdx.x;
  // should access row i
  hipMemcpy(trace[i], accessor[i]);
}
*/
__global__ void packed_accessor_kernel(
    torch::PackedTensorAccessor32<float, 3> accessor,
    float** trace, int size) {
  int i=threadIdx.x;
  // should access row i
  for (int j = 0; j < size; j++) {
    //hipMemcpy(&trace[i][j], accessor[i][j].data(), sizeof(float), hipMemcpyDeviceToDevice);
    trace[i][j] = *(accessor[i][j].data());
    
  }
}

void cublas_bmm_wrapper_accessor(hipblasHandle_t handle,
               torch::Tensor d_A, torch::Tensor d_B, torch::Tensor d_C,
               size_t a_rows, size_t b_cols, size_t b_rows,
               size_t batch_dim) {


    size_t c_size = sizeof(float) * a_rows * b_cols;
    size_t a_size = sizeof(float) * a_rows * b_rows;
    size_t b_size = sizeof(float) * b_rows * b_cols;

    float **d_A_arr, **d_B_arr;
    float **d_C_arr;

    gpuErrchk(hipMalloc((void **)&d_A_arr, batch_dim * sizeof(float *)));
    gpuErrchk(hipMalloc((void **)&d_B_arr, batch_dim * sizeof(float *)));
    gpuErrchk(hipMalloc((void **)&d_C_arr, batch_dim * sizeof(float *)));

    auto A_accessor = d_A.packed_accessor32<float,3>();
    float **trace = d_A_arr;
    // execute 1 time with a_rows threads
    packed_accessor_kernel<<<1, a_rows>>>(A_accessor, (float**) (d_A_arr), a_size);

    const float alpha = 1.0f, beta = 0.0f;
    hipblasStatus_t cublas_result = hipblasSgemmBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N
                                       , b_cols, a_rows, b_rows
                                       , &alpha, d_B_arr, b_cols, d_A_arr, b_rows
                                       , &beta, d_C_arr, b_cols
                                       , batch_dim);
    assert(cublas_result == HIPBLAS_STATUS_SUCCESS);

    gpuErrchk(hipFree(d_A_arr));
    gpuErrchk(hipFree(d_B_arr));
    gpuErrchk(hipFree(d_C_arr));
}


void cublas_bmm_wrapper(hipblasHandle_t handle,
               float *d_A, float *d_B, float *d_C,
               size_t a_rows, size_t b_cols, size_t b_rows,
               size_t batch_dim) {


    size_t c_size = sizeof(float) * a_rows * b_cols;
    size_t a_size = sizeof(float) * a_rows * b_rows;
    size_t b_size = sizeof(float) * b_rows * b_cols;

    float **d_A_arr, **d_B_arr;
    float **d_C_arr;

    gpuErrchk(hipMalloc((void **)&d_A_arr, batch_dim * sizeof(float *)));
    gpuErrchk(hipMalloc((void **)&d_B_arr, batch_dim * sizeof(float *)));
    gpuErrchk(hipMalloc((void **)&d_C_arr, batch_dim * sizeof(float *)));

    // assumes contiguous memory
    for (int i = 0; i < batch_dim; i++) {
        gpuErrchk(hipMemcpy(d_A_arr[i], d_A + (batch_dim * a_size * i), (batch_dim * a_size), hipMemcpyDeviceToDevice));
        gpuErrchk(hipMemcpy(d_B_arr[i], d_B + (batch_dim * b_size * i), (batch_dim * b_size), hipMemcpyDeviceToDevice));
        gpuErrchk(hipMemcpy(d_C_arr[i], d_C + (batch_dim * c_size * i), (batch_dim * c_size), hipMemcpyDeviceToDevice));
    }

    const float alpha = 1.0f, beta = 0.0f;
    hipblasStatus_t cublas_result = hipblasSgemmBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N
                                       , b_cols, a_rows, b_rows
                                       , &alpha, d_B_arr, b_cols, d_A_arr, b_rows
                                       , &beta, d_C_arr, b_cols
                                       , batch_dim);
    assert(cublas_result == HIPBLAS_STATUS_SUCCESS);

    gpuErrchk(hipFree(d_A_arr));
    gpuErrchk(hipFree(d_B_arr));
    gpuErrchk(hipFree(d_C_arr));
}

// https://stackoverflow.com/questions/29688627/sparse-matrix-matrix-multiplication-in-cuda-using-cusparse

// Sparse (CSR) * Dense matmul
// A * B = C

// (m x k) * (k * n) = (m x n)
// note: row_ind.len = lda + 1

void cusparse_mm_wrapper(hipsparseHandle_t handle,
                         double *h_A, int *h_A_ColIndices, int *h_A_RowIndices,
                         int nnzA, int h_A_rowptr_size,
                         double *h_B_dense, int h_B_rows, int h_B_cols,
                         double *h_C_dense)
{
    // Initialize cuSPARSE
    // hipsparseHandle_t handle;
    // cusparseSafeCall(hipsparseCreate(&handle));
    const int m = h_A_rowptr_size - 1;
    const int k = h_B_rows;
    const int n = h_B_cols;

    // Host side dense matrices
    //double *h_C_dense = (double *)malloc(m * n * sizeof(*h_C_dense));
    // Create device arrays and copy host arrays to them
    double *d_B_dense;
    gpuErrchk(hipMalloc(&d_B_dense, k * n * sizeof(*d_B_dense)));
    double *d_C_dense;
    gpuErrchk(hipMalloc(&d_C_dense, m * n * sizeof(*d_C_dense)));

    // copy B from host to device
    gpuErrchk(hipMemcpy(d_B_dense, h_B_dense, k * n * sizeof(*d_B_dense), hipMemcpyHostToDevice));

    // Descriptor for sparse matrix A
    hipsparseMatDescr_t descrA;
    cusparseSafeCall(hipsparseCreateMatDescr(&descrA));
    cusparseSafeCall(hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL));
    cusparseSafeCall(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ONE));
    // Descriptor for sparse matrix B
    hipsparseMatDescr_t descrB;
    cusparseSafeCall(hipsparseCreateMatDescr(&descrB));
    cusparseSafeCall(hipsparseSetMatType(descrB, HIPSPARSE_MATRIX_TYPE_GENERAL));
    cusparseSafeCall(hipsparseSetMatIndexBase(descrB, HIPSPARSE_INDEX_BASE_ONE));
    // Descriptor for sparse matrix C
    hipsparseMatDescr_t descrC;
    cusparseSafeCall(hipsparseCreateMatDescr(&descrC));
    cusparseSafeCall(hipsparseSetMatType(descrC, HIPSPARSE_MATRIX_TYPE_GENERAL));
    cusparseSafeCall(hipsparseSetMatIndexBase(descrC, HIPSPARSE_INDEX_BASE_ONE));

    int nnzB = 0; //   Number of nonzero elements in dense matrix B
    // Device side number of nonzero elements per row of matrix B
    int *d_nnzPerVectorB;
    gpuErrchk(hipMalloc(&d_nnzPerVectorB, k * sizeof(*d_nnzPerVectorB)));
    cusparseSafeCall(hipsparseDnnz(handle, HIPSPARSE_DIRECTION_ROW, k, n, descrB, d_B_dense, k, d_nnzPerVectorB, &nnzB));
    // Host side number of nonzero elements per row of matrix B
    int *h_nnzPerVectorB = (int *)malloc(k * sizeof(*h_nnzPerVectorB));
    gpuErrchk(hipMemcpy(h_nnzPerVectorB, d_nnzPerVectorB, k * sizeof(*h_nnzPerVectorB), hipMemcpyDeviceToHost));

    // Device side sparse matrix A
    double *d_A;
    gpuErrchk(hipMalloc(&d_A, nnzA * sizeof(*d_A)));
    int *d_A_RowIndices;
    gpuErrchk(hipMalloc(&d_A_RowIndices, (m + 1) * sizeof(*d_A_RowIndices)));
    int *d_A_ColIndices;
    gpuErrchk(hipMalloc(&d_A_ColIndices, nnzA * sizeof(*d_A_ColIndices)));
    // Copy A from host to device
    hipMemcpy(d_A, h_A, nnzA * sizeof(*d_A), hipMemcpyHostToDevice);
    hipMemcpy(d_A_RowIndices, h_A_RowIndices, (m + 1) * sizeof(*d_A_RowIndices), hipMemcpyHostToDevice);
    hipMemcpy(d_A_ColIndices, h_A_ColIndices, nnzA * sizeof(*d_A_ColIndices), hipMemcpyHostToDevice);

    // Device side sparse matrix B
    double *d_B;
    gpuErrchk(hipMalloc(&d_B, nnzB * sizeof(*d_B)));
    int *d_B_RowIndices;
    gpuErrchk(hipMalloc(&d_B_RowIndices, (k + 1) * sizeof(*d_B_RowIndices)));
    int *d_B_ColIndices;
    gpuErrchk(hipMalloc(&d_B_ColIndices, nnzB * sizeof(*d_B_ColIndices)));
    // Dense B to Sparse B
    cusparseSafeCall(hipsparseDdense2csr(handle, k, n, descrB, d_B_dense, k, d_nnzPerVectorB, d_B, d_B_RowIndices, d_B_ColIndices));

    // Move sparse B from device to host
    double *h_B = (double *)malloc(nnzB * sizeof(*h_B));
    int *h_B_ColIndices = (int *)malloc(nnzB * sizeof(*h_B_ColIndices));
    int *h_B_RowIndices = (int *)malloc((k + 1) * sizeof(*h_B_RowIndices));

    gpuErrchk(hipMemcpy(h_B, d_B, nnzB * sizeof(*h_B), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_B_RowIndices, d_B_RowIndices, (k + 1) * sizeof(*h_B_RowIndices), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_B_ColIndices, d_B_ColIndices, nnzB * sizeof(*h_B_ColIndices), hipMemcpyDeviceToHost));

    // Device side sparse matrix C
    int *d_C_RowIndices;
    gpuErrchk(hipMalloc(&d_C_RowIndices, (m + 1) * sizeof(*d_C_RowIndices)));

    // Performing the matrix - matrix multiplication
    int baseC, nnzC = 0;
    // nnzTotalDevHostPtr points to host memory
    int *nnzTotalDevHostPtr = &nnzC;

    cusparseSafeCall(hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST));

    cusparseSafeCall(hipsparseXcsrgemmNnz(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, n, k, descrA, nnzA,
                                         d_A_RowIndices, d_A_ColIndices, descrB, nnzB, d_B_RowIndices, d_B_ColIndices, descrC, d_C_RowIndices,
                                         nnzTotalDevHostPtr));
    if (nnzTotalDevHostPtr != NULL)
        nnzC = *nnzTotalDevHostPtr;
    else
    {
        hipMemcpy(&nnzC, d_C_RowIndices + m, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&baseC, d_C_RowIndices, sizeof(int), hipMemcpyDeviceToHost);
        nnzC -= baseC;
    }

    // device side sparse matrix C
    double *d_C;
    gpuErrchk(hipMalloc(&d_C, nnzC * sizeof(double)));
    int *d_C_ColIndices;
    gpuErrchk(hipMalloc(&d_C_ColIndices, nnzC * sizeof(int)));
    // host side sparse matrix c
    double *h_C = (double *)malloc(nnzC * sizeof(*h_C));
    int *h_C_ColIndices = (int *)malloc(nnzC * sizeof(*h_C_ColIndices));
    int *h_C_RowIndices = (int *)malloc((m + 1) * sizeof(*h_C_RowIndices));

    cusparseSafeCall(hipsparseDcsrgemm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, n, k, descrA, nnzA,
                                      d_A, d_A_RowIndices, d_A_ColIndices, descrB, nnzB, d_B, d_B_RowIndices, d_B_ColIndices, descrC,
                                      d_C, d_C_RowIndices, d_C_ColIndices));

    cusparseSafeCall(hipsparseDcsr2dense(handle, m, n, descrC, d_C, d_C_RowIndices, d_C_ColIndices, d_C_dense, m));

    gpuErrchk(hipMemcpy(h_C, d_C, nnzC * sizeof(*h_C), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_C_RowIndices, d_C_RowIndices, (m + 1) * sizeof(*h_C_RowIndices), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_C_ColIndices, d_C_ColIndices, nnzC * sizeof(*h_C_ColIndices), hipMemcpyDeviceToHost));

    gpuErrchk(hipMemcpy(h_C_dense, d_C_dense, m * n * sizeof(double), hipMemcpyDeviceToHost));

    /*
    for (int j = 0; j < m; j++)
    {
        for (int i = 0; i < n; i++)
            printf("%f \t", h_C_dense[i * m + j]);
        printf("\n");
    }
    */

    // free(h_C_dense);
    hipFree(h_B_dense);
    hipFree(h_C_dense);

    hipFree(d_nnzPerVectorB);
    free(h_nnzPerVectorB);

    hipFree(d_A);
    hipFree(d_A_RowIndices);
    hipFree(d_A_ColIndices);

    hipFree(d_B);
    hipFree(d_B_RowIndices);
    hipFree(d_B_ColIndices);

    hipFree(d_C);
    hipFree(d_C_RowIndices);
    hipFree(d_C_ColIndices);

    free(h_C);
    free(h_C_RowIndices);
    free(h_C_ColIndices);

    return;
}

void dense_to_csr(hipsparseHandle_t handle, 
                  double *h_A_dense, const int Nrows, const int Ncols,
                  double **h_A_val, int **h_A_colind, int **h_A_rowptr, int *nnzA)
{
    // Initialize cuSPARSE
    // hipsparseHandle_t handle;
    // cusparseSafeCall(hipsparseCreate(&handle));

    //create device array and copy host to it
    double *d_A_dense;
    gpuErrchk(hipMalloc(&d_A_dense, Nrows * Ncols * sizeof(*d_A_dense)));
    gpuErrchk(hipMemcpy(d_A_dense, h_A_dense, Nrows * Ncols * sizeof(*d_A_dense), hipMemcpyHostToDevice));

    // Descriptor for sparse matrix A
    hipsparseMatDescr_t descrA;
    cusparseSafeCall(hipsparseCreateMatDescr(&descrA));
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);

    int nnz = 0;           //   Number of nonzero elements in dense matrix
    const int lda = Nrows; //   Leading dimension of dense matrix
    // Device side number of nonzero elements per row
    int *d_nnzPerVector;
    gpuErrchk(hipMalloc(&d_nnzPerVector, Nrows * sizeof(*d_nnzPerVector)));
    cusparseSafeCall(hipsparseDnnz(handle, HIPSPARSE_DIRECTION_ROW, Nrows, Ncols, descrA, d_A_dense, lda, d_nnzPerVector, &nnz));
    // Host side number of nonzero elements per row
    int *h_nnzPerVector = (int *)malloc(Nrows * sizeof(*h_nnzPerVector));
    gpuErrchk(hipMemcpy(h_nnzPerVector, d_nnzPerVector, Nrows * sizeof(*h_nnzPerVector), hipMemcpyDeviceToHost));

    // Device side dense matrix
    double *d_A;
    gpuErrchk(hipMalloc(&d_A, nnz * sizeof(*d_A)));
    int *d_A_RowIndices;
    gpuErrchk(hipMalloc(&d_A_RowIndices, (Nrows + 1) * sizeof(*d_A_RowIndices)));
    int *d_A_ColIndices;
    gpuErrchk(hipMalloc(&d_A_ColIndices, nnz * sizeof(*d_A_ColIndices)));

    cusparseSafeCall(hipsparseDdense2csr(handle, Nrows, Ncols, descrA, d_A_dense, lda, d_nnzPerVector, d_A, d_A_RowIndices, d_A_ColIndices));

    // Host side dense matrix
    double *h_A = (double *)malloc(nnz * sizeof(*h_A));
    int *h_A_RowIndices = (int *)malloc((Nrows + 1) * sizeof(*h_A_RowIndices));
    int *h_A_ColIndices = (int *)malloc(nnz * sizeof(*h_A_ColIndices));
    gpuErrchk(hipMemcpy(h_A, d_A, nnz * sizeof(*h_A), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_A_RowIndices, d_A_RowIndices, (Nrows + 1) * sizeof(*h_A_RowIndices), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_A_ColIndices, d_A_ColIndices, nnz * sizeof(*h_A_ColIndices), hipMemcpyDeviceToHost));

    *h_A_val = h_A;
    *h_A_rowptr = h_A_RowIndices;
    *h_A_colind = h_A_ColIndices;
    *nnzA = nnz;

    gpuErrchk(hipFree(d_nnzPerVector));
    free(h_nnzPerVector);

    gpuErrchk(hipFree(d_A));
    gpuErrchk(hipFree(d_A_RowIndices));
    gpuErrchk(hipFree(d_A_ColIndices));

    return;
}






// void cublas_bmm_wrapper(hipblasHandle_t handle,
//                float **A, float **B, float **C,
//                size_t a_rows, size_t b_cols, size_t b_rows,
//                size_t batch_dim) {

//     float *d_A[batch_dim];
//     float *d_B[batch_dim];
//     float *d_C[batch_dim];

//     size_t c_size = sizeof(float) * a_rows * b_cols;
//     size_t a_size = sizeof(float) * a_rows * b_rows;
//     size_t b_size = sizeof(float) * b_rows * b_cols;

//     const float **d_A_arr, **d_B_arr;
//     float **d_C_arr;

//     for (int i = 0; i < batch_dim; i++) {
//         gpuErrchk(hipMalloc((void **)&d_A[i], a_size));
//         gpuErrchk(hipMalloc((void **)&d_B[i], b_size));
//         gpuErrchk(hipMalloc((void **)&d_C[i], c_size));
//     }

//     gpuErrchk(hipMalloc((void **)&d_A_arr, batch_dim * sizeof(float *)));
//     gpuErrchk(hipMalloc((void **)&d_B_arr, batch_dim * sizeof(float *)));
//     gpuErrchk(hipMalloc((void **)&d_C_arr, batch_dim * sizeof(float *)));

//     for (int i = 0; i < batch_dim; i++) {
//         gpuErrchk(hipMemcpy(d_A[i], A[i], a_size, hipMemcpyHostToDevice));
//         gpuErrchk(hipMemcpy(d_B[i], B[i], b_size, hipMemcpyHostToDevice));
//         gpuErrchk(hipMemcpy(d_C[i], C[i], c_size, hipMemcpyHostToDevice));
//     }

//     gpuErrchk(hipMemcpy(d_A_arr, d_A, batch_dim * sizeof(float *), hipMemcpyHostToDevice));
//     gpuErrchk(hipMemcpy(d_B_arr, d_B, batch_dim * sizeof(float *), hipMemcpyHostToDevice));
//     gpuErrchk(hipMemcpy(d_C_arr, d_C, batch_dim * sizeof(float *), hipMemcpyHostToDevice));

//     const float alpha = 1.0f, beta = 0.0f;
//     hipblasStatus_t cublas_result = hipblasSgemmBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N
//                                        , b_cols, a_rows, b_rows
//                                        , &alpha, d_B_arr, b_cols, d_A_arr, b_rows
//                                        , &beta, d_C_arr, b_cols
//                                        , batch_dim);
//     assert(cublas_result == HIPBLAS_STATUS_SUCCESS);

//     for (int i = 0; i < batch_dim; i++)
//     {
//         gpuErrchk(hipMemcpy(C[i], d_C[i], c_size, hipMemcpyDeviceToHost));
//         gpuErrchk(hipFree(d_A[i]));
//         gpuErrchk(hipFree(d_B[i]));
//         gpuErrchk(hipFree(d_C[i]));
//     }
//     gpuErrchk(hipFree(d_A_arr));
//     gpuErrchk(hipFree(d_B_arr));
//     gpuErrchk(hipFree(d_C_arr));
// }

#endif // __CUBLAS_BMM_KERNEL_H__
